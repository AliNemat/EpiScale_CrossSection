#include "hip/hip_runtime.h"
#include "SceNodes.h"

__constant__ double sceInterPara[5];
__constant__ double sceIntraPara[5];
// parameter set for cells that are going to divide
__constant__ double sceIntraParaDiv[5];
__constant__ double sceDivProPara;
__constant__ double sceCartPara[5];
__constant__ double sceInterDiffPara[5];
__constant__ double sceProfilePara[7];
__constant__ double sceECMPara[5];
__constant__ double sceDiffPara[5];

__constant__ double cartGrowDirVec[3];
__constant__ uint ProfilebeginPos;
__constant__ uint ECMbeginPos;
__constant__ uint cellNodeBeginPos;
__constant__ uint nodeCountPerECM;
__constant__ uint nodeCountPerCell;
//
//
__constant__ uint cellNodeBeginPos_M;
__constant__ uint allNodeCountPerCell_M;
__constant__ uint bdryThreshold_M;
__constant__ double sceInterPara_M[5];
__constant__ double sceIntraBPara_M[5];
__constant__ double maxAdhBondLength;
__constant__ double bondStiff;
__constant__ double bondAdhThreshold;
__constant__ double minAdhBondLength;

// This template method expands an input sequence by
// replicating each element a variable number of times. For example,
//
//   expand([2,2,2],[A,B,C]) -> [A,A,B,B,C,C]
//   expand([3,0,1],[A,B,C]) -> [A,A,A,C]
//   expand([1,3,2],[A,B,C]) -> [A,B,B,B,C,C]
//
// The element counts are assumed to be non-negative integers
template<typename InputIterator1, typename InputIterator2,
		typename OutputIterator>
OutputIterator expand(InputIterator1 first1, InputIterator1 last1,
		InputIterator2 first2, OutputIterator output) {
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;

	difference_type input_size = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin());

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if(thrust::counting_iterator<difference_type>(0),
			thrust::counting_iterator<difference_type>(input_size),
			output_offsets.begin(), first1, output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan(output_indices.begin(), output_indices.end(),
			output_indices.begin(), thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output;
	thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(), output_indices.end(), first2,
			output);

	// return output + output_size
	thrust::advance(output, output_size);
	return output;
}

SceNodes::SceNodes() {
	readDomainPara();
}

void SceNodes::readDomainPara() {
	domainPara.minX = globalConfigVars.getConfigValue("DOMAIN_XMIN").toDouble();
	domainPara.maxX = globalConfigVars.getConfigValue("DOMAIN_XMAX").toDouble();
	domainPara.minY = globalConfigVars.getConfigValue("DOMAIN_YMIN").toDouble();
	domainPara.maxY = globalConfigVars.getConfigValue("DOMAIN_YMAX").toDouble();
	domainPara.minZ = globalConfigVars.getConfigValue("DOMAIN_ZMIN").toDouble();
	domainPara.maxZ = globalConfigVars.getConfigValue("DOMAIN_ZMAX").toDouble();
	domainPara.gridSpacing = getMaxEffectiveRange();
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInYDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
}

void SceNodes::readMechPara() {
	double U0 =
			globalConfigVars.getConfigValue("InterCell_U0_Original").toDouble()
					/ globalConfigVars.getConfigValue("InterCell_U0_DivFactor").toDouble();
	double V0 =
			globalConfigVars.getConfigValue("InterCell_V0_Original").toDouble()
					/ globalConfigVars.getConfigValue("InterCell_V0_DivFactor").toDouble();
	double k1 =
			globalConfigVars.getConfigValue("InterCell_k1_Original").toDouble()
					/ globalConfigVars.getConfigValue("InterCell_k1_DivFactor").toDouble();
	double k2 =
			globalConfigVars.getConfigValue("InterCell_k2_Original").toDouble()
					/ globalConfigVars.getConfigValue("InterCell_k2_DivFactor").toDouble();
	double interLinkEffectiveRange = globalConfigVars.getConfigValue(
			"InterCellLinkEffectRange").toDouble();

	mechPara.sceInterParaCPU[0] = U0;
	mechPara.sceInterParaCPU[1] = V0;
	mechPara.sceInterParaCPU[2] = k1;
	mechPara.sceInterParaCPU[3] = k2;
	mechPara.sceInterParaCPU[4] = interLinkEffectiveRange;

	double U0_Intra =
			globalConfigVars.getConfigValue("IntraCell_U0_Original").toDouble()
					/ globalConfigVars.getConfigValue("IntraCell_U0_DivFactor").toDouble();
	double V0_Intra =
			globalConfigVars.getConfigValue("IntraCell_V0_Original").toDouble()
					/ globalConfigVars.getConfigValue("IntraCell_V0_DivFactor").toDouble();
	double k1_Intra =
			globalConfigVars.getConfigValue("IntraCell_k1_Original").toDouble()
					/ globalConfigVars.getConfigValue("IntraCell_k1_DivFactor").toDouble();
	double k2_Intra =
			globalConfigVars.getConfigValue("IntraCell_k2_Original").toDouble()
					/ globalConfigVars.getConfigValue("IntraCell_k2_DivFactor").toDouble();
	double intraLinkEffectiveRange = globalConfigVars.getConfigValue(
			"IntraCellLinkEffectRange").toDouble();

	mechPara.sceIntraParaCPU[0] = U0_Intra;
	mechPara.sceIntraParaCPU[1] = V0_Intra;
	mechPara.sceIntraParaCPU[2] = k1_Intra;
	mechPara.sceIntraParaCPU[3] = k2_Intra;
	mechPara.sceIntraParaCPU[4] = intraLinkEffectiveRange;

	if (controlPara.simuType == Beak) {

		double U0_Cart =
				globalConfigVars.getConfigValue("InterCell_U0_Original").toDouble()
						/ globalConfigVars.getConfigValue("Cart_U0_DivFactor").toDouble();
		double V0_Cart =
				globalConfigVars.getConfigValue("InterCell_V0_Original").toDouble()
						/ globalConfigVars.getConfigValue("Cart_V0_DivFactor").toDouble();
		double k1_Cart =
				globalConfigVars.getConfigValue("InterCell_k1_Original").toDouble()
						/ globalConfigVars.getConfigValue("Cart_k1_DivFactor").toDouble();
		double k2_Cart =
				globalConfigVars.getConfigValue("InterCell_k2_Original").toDouble()
						/ globalConfigVars.getConfigValue("Cart_k2_DivFactor").toDouble();
		double cartProfileEffectiveRange = globalConfigVars.getConfigValue(
				"CartForceEffectiveRange").toDouble();
		mechPara.sceCartParaCPU[0] = U0_Cart;
		mechPara.sceCartParaCPU[1] = V0_Cart;
		mechPara.sceCartParaCPU[2] = k1_Cart;
		mechPara.sceCartParaCPU[3] = k2_Cart;
		mechPara.sceCartParaCPU[4] = cartProfileEffectiveRange;

		// 1.8 comes from standard
		double neutralLength = globalConfigVars.getConfigValue(
				"Epi_link_neutral_dist").toDouble();

		double linearParameter = globalConfigVars.getConfigValue(
				"Epi_linear_parameter").toDouble();

		double U0_Bdry =
				globalConfigVars.getConfigValue("InterCell_U0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Bdry_U0_DivFactor").toDouble();
		double V0_Bdry =
				globalConfigVars.getConfigValue("InterCell_V0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Bdry_V0_DivFactor").toDouble();
		double k1_Bdry =
				globalConfigVars.getConfigValue("InterCell_k1_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Bdry_k1_DivFactor").toDouble();
		double k2_Bdry =
				globalConfigVars.getConfigValue("InterCell_k2_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Bdry_k2_DivFactor").toDouble();

		mechPara.sceProfileParaCPU[0] = U0_Bdry;
		mechPara.sceProfileParaCPU[1] = V0_Bdry;
		mechPara.sceProfileParaCPU[2] = k1_Bdry;
		mechPara.sceProfileParaCPU[3] = k2_Bdry;
		mechPara.sceProfileParaCPU[4] = interLinkEffectiveRange;
		mechPara.sceProfileParaCPU[5] = linearParameter;
		mechPara.sceProfileParaCPU[6] = neutralLength;

		double U0_ECM =
				globalConfigVars.getConfigValue("InterCell_U0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_ECM_U0_DivFactor").toDouble();
		double V0_ECM =
				globalConfigVars.getConfigValue("InterCell_V0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_ECM_V0_DivFactor").toDouble();
		double k1_ECM =
				globalConfigVars.getConfigValue("InterCell_k1_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_ECM_k1_DivFactor").toDouble();
		double k2_ECM =
				globalConfigVars.getConfigValue("InterCell_k2_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_ECM_k2_DivFactor").toDouble();
		mechPara.sceECMParaCPU[0] = U0_ECM;
		mechPara.sceECMParaCPU[1] = V0_ECM;
		mechPara.sceECMParaCPU[2] = k1_ECM;
		mechPara.sceECMParaCPU[3] = k2_ECM;
		mechPara.sceECMParaCPU[4] = interLinkEffectiveRange;
		double U0_Diff =
				globalConfigVars.getConfigValue("InterCell_U0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Diff_U0_DivFactor").toDouble();
		double V0_Diff =
				globalConfigVars.getConfigValue("InterCell_V0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Diff_V0_DivFactor").toDouble();
		double k1_Diff =
				globalConfigVars.getConfigValue("InterCell_k1_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Diff_k1_DivFactor").toDouble();
		double k2_Diff =
				globalConfigVars.getConfigValue("InterCell_k2_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"InterCell_Diff_k2_DivFactor").toDouble();

		mechPara.sceInterDiffParaCPU[0] = U0_Diff;
		mechPara.sceInterDiffParaCPU[1] = V0_Diff;
		mechPara.sceInterDiffParaCPU[2] = k1_Diff;
		mechPara.sceInterDiffParaCPU[3] = k2_Diff;
		mechPara.sceInterDiffParaCPU[4] = interLinkEffectiveRange;

	} else if (controlPara.simuType == Disc) {
		double U0_Intra_Div =
				globalConfigVars.getConfigValue("IntraCell_U0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"IntraCell_U0_Div_DivFactor").toDouble();
		double V0_Intra_Div =
				globalConfigVars.getConfigValue("IntraCell_V0_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"IntraCell_V0_Div_DivFactor").toDouble();
		double k1_Intra_Div =
				globalConfigVars.getConfigValue("IntraCell_k1_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"IntraCell_k1_Div_DivFactor").toDouble();
		double k2_Intra_Div =
				globalConfigVars.getConfigValue("IntraCell_k2_Original").toDouble()
						/ globalConfigVars.getConfigValue(
								"IntraCell_k2_Div_DivFactor").toDouble();
		double growthProgressThreshold = globalConfigVars.getConfigValue(
				"GrowthProgressThreshold").toDouble();

		mechPara.sceIntraParaDivCPU[0] = U0_Intra_Div;
		mechPara.sceIntraParaDivCPU[1] = V0_Intra_Div;
		mechPara.sceIntraParaDivCPU[2] = k1_Intra_Div;
		mechPara.sceIntraParaDivCPU[3] = k2_Intra_Div;
		mechPara.sceIntraParaDivCPU[4] = growthProgressThreshold;
	}
}

SceNodes::SceNodes(uint totalBdryNodeCount, uint maxProfileNodeCount,
		uint maxCartNodeCount, uint maxTotalECMCount, uint maxNodeInECM,
		uint maxTotalCellCount, uint maxNodeInCell, bool isStab) {
	initControlPara(isStab);
	readDomainPara();
	uint maxTotalNodeCount;
	if (controlPara.simuType != Disc_M) {
		initNodeAllocPara(totalBdryNodeCount, maxProfileNodeCount,
				maxCartNodeCount, maxTotalECMCount, maxNodeInECM,
				maxTotalCellCount, maxNodeInCell);
		maxTotalNodeCount = totalBdryNodeCount + maxProfileNodeCount
				+ maxCartNodeCount + allocPara.maxTotalECMNodeCount
				+ allocPara.maxTotalCellNodeCount;
	} else {
		uint maxEpiNodeCount = globalConfigVars.getConfigValue(
				"MaxEpiNodeCountPerCell").toInt();
		uint maxInternalNodeCount = globalConfigVars.getConfigValue(
				"MaxAllNodeCountPerCell").toInt() - maxEpiNodeCount;

		initNodeAllocPara_M(totalBdryNodeCount, maxTotalCellCount,
				maxEpiNodeCount, maxInternalNodeCount);
		maxTotalNodeCount = allocPara_M.maxTotalNodeCount;
	}
	allocSpaceForNodes(maxTotalNodeCount);
	thrust::host_vector<SceNodeType> hostTmpVector(maxTotalNodeCount);
	thrust::host_vector<bool> hostTmpVector2(maxTotalNodeCount);
	thrust::host_vector<int> hostTmpVector3(maxTotalNodeCount);

	if (controlPara.simuType != Disc_M) {

		for (int i = 0; i < maxTotalNodeCount; i++) {
			if (i < allocPara.startPosProfile) {
				hostTmpVector[i] = Boundary;
				hostTmpVector3[i] = 0;
			} else if (i < allocPara.startPosCart) {
				hostTmpVector[i] = Profile;
				hostTmpVector3[i] = 0;
			} else if (i < allocPara.startPosECM) {
				hostTmpVector[i] = Cart;
				hostTmpVector3[i] = 0;
			} else if (i < allocPara.startPosCells) {
				hostTmpVector[i] = ECM;
				hostTmpVector3[i] = (i - allocPara.startPosECM)
						/ allocPara.maxNodePerECM;
			} else {
				// all initialized as FNM
				hostTmpVector[i] = FNM;
				hostTmpVector3[i] = (i - allocPara.startPosCells)
						/ allocPara.maxNodeOfOneCell;
			}
			hostTmpVector2[i] = false;
		}

	} else {
		for (uint i = 0; i < maxTotalNodeCount; i++) {
			if (i < allocPara_M.bdryNodeCount) {
				hostTmpVector[i] = Boundary;
				hostTmpVector3[i] = 0;
			} else {
				uint tmp = i - allocPara_M.bdryNodeCount;
				uint cellRank = tmp / allocPara_M.bdryNodeCount;
				uint nodeRank = tmp % allocPara_M.bdryNodeCount;
				if (nodeRank < allocPara_M.maxMembrNodePerCell) {
					hostTmpVector[i] = CellMembr;
				} else {
					hostTmpVector[i] = CellIntnl;
				}
				hostTmpVector3[i] = cellRank;
			}
			hostTmpVector2[i] = false;
		}
	}
	infoVecs.nodeCellType = hostTmpVector;
	infoVecs.nodeIsActive = hostTmpVector2;
	infoVecs.nodeCellRank = hostTmpVector3;

	copyParaToGPUConstMem();
}

SceNodes::SceNodes(uint maxTotalCellCount, uint maxAllNodePerCell) {
	//initControlPara (isStab);
	int simuTypeConfigValue =
			globalConfigVars.getConfigValue("SimulationType").toInt();
	controlPara.simuType = parseTypeFromConfig(simuTypeConfigValue);
	readDomainPara();
	uint maxTotalNodeCount = maxTotalCellCount * maxAllNodePerCell;

	uint maxMembrNodeCountPerCell = globalConfigVars.getConfigValue(
			"MaxMembrNodeCountPerCell").toInt();
	uint maxIntnlNodeCountPerCell = globalConfigVars.getConfigValue(
			"MaxIntnlNodeCountPerCell").toInt();

	initNodeAllocPara_M(0, maxTotalCellCount, maxMembrNodeCountPerCell,
			maxIntnlNodeCountPerCell);

	std::cout << "bdry node count = " << allocPara_M.bdryNodeCount << std::endl;
	std::cout << "max cell count = " << allocPara_M.maxCellCount << std::endl;
	std::cout << "max node per cell = " << allocPara_M.maxAllNodePerCell
			<< std::endl;
	std::cout << "max membr node per cell= " << allocPara_M.maxMembrNodePerCell
			<< std::endl;
	std::cout << "max intnl node per cell= " << allocPara_M.maxIntnlNodePerCell
			<< std::endl;
	std::cout << "max total node count= " << allocPara_M.maxTotalNodeCount
			<< std::endl;

	allocSpaceForNodes(maxTotalNodeCount);
	thrust::host_vector<SceNodeType> hostTmpVector(maxTotalNodeCount);
	thrust::host_vector<bool> hostTmpVector2(maxTotalNodeCount);

	uint nodeRank;
	for (uint i = 0; i < maxTotalNodeCount; i++) {
		if (i < allocPara_M.bdryNodeCount) {
			hostTmpVector[i] = Boundary;
		} else {
			uint tmp = i - allocPara_M.bdryNodeCount;
			nodeRank = tmp % allocPara_M.maxAllNodePerCell;
			if (nodeRank < allocPara_M.maxMembrNodePerCell) {
				hostTmpVector[i] = CellMembr;
				//std::cout << "0";
			} else {
				hostTmpVector[i] = CellIntnl;
				//std::cout << "1";
			}

		}
		hostTmpVector2[i] = false;
		if (nodeRank == 0) {
			//std::cout << std::endl;
		}
	}
	//std::cout << "finished" << std::endl;
	//std::cout.flush();
	infoVecs.nodeCellType = hostTmpVector;
	infoVecs.nodeIsActive = hostTmpVector2;

	thrust::host_vector<int> bondVec(maxTotalNodeCount, -1);
	infoVecs.nodeAdhereIndex = bondVec;
	//std::cout << "copy finished!" << std::endl;
	//std::cout.flush();
	//copyParaToGPUConstMem();
	//std::cout << "at the end" << std::endl;
	//std::cout.flush();
}

void SceNodes::copyParaToGPUConstMem() {

	readMechPara();

	hipMemcpyToSymbol(HIP_SYMBOL(sceInterPara), mechPara.sceInterParaCPU,
			5 * sizeof(double));

	hipMemcpyToSymbol(HIP_SYMBOL(sceIntraPara), mechPara.sceIntraParaCPU,
			5 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sceIntraParaDiv), mechPara.sceIntraParaDivCPU,
			5 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(ProfilebeginPos), &allocPara.startPosProfile,
			sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(ECMbeginPos), &allocPara.startPosECM, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(cellNodeBeginPos), &allocPara.startPosCells,
			sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(nodeCountPerECM), &allocPara.maxNodePerECM, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(nodeCountPerCell), &allocPara.maxNodeOfOneCell,
			sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(sceCartPara), mechPara.sceCartParaCPU,
			5 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sceProfilePara), mechPara.sceProfileParaCPU,
			7 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sceInterDiffPara), mechPara.sceInterDiffParaCPU,
			5 * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(sceECMPara), mechPara.sceECMParaCPU, 5 * sizeof(double));

}

void SceNodes::initDimension(double domainMinX, double domainMaxX,
		double domainMinY, double domainMaxY, double domainBucketSize) {
	domainPara.minX = domainMinX;
	domainPara.maxX = domainMaxX;
	domainPara.minY = domainMinY;
	domainPara.maxY = domainMaxY;
	domainPara.gridSpacing = domainBucketSize;
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInYDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
	domainPara.totalBucketCount = domainPara.numOfBucketsInXDim
			* domainPara.numOfBucketsInYDim;

	auxVecs.keyBegin.resize(domainPara.totalBucketCount);
	auxVecs.keyEnd.resize(domainPara.totalBucketCount);
}

std::vector<std::pair<uint, uint> > SceNodes::obtainPossibleNeighborPairs() {
	std::vector<std::pair<uint, uint> > result;
	thrust::host_vector<uint> keyBeginCPU = auxVecs.keyBegin;
	thrust::host_vector<uint> keyEndCPU = auxVecs.keyEnd;
	thrust::host_vector<uint> bucketKeysCPU = auxVecs.bucketKeys;
	thrust::host_vector<uint> bucketValuesCPU = auxVecs.bucketValues;
	thrust::host_vector<uint> bucketValuesExtendedCPU =
			auxVecs.bucketValuesIncludingNeighbor;
	uint iterationCounter = 0;
	int size = bucketKeysCPU.size();
	for (int i = 0; i < size; i++) {
		for (int j = keyBeginCPU[bucketKeysCPU[i]];
				j < keyEndCPU[bucketKeysCPU[i]]; j++) {
			int node1 = bucketValuesCPU[i];
			int node2 = bucketValuesExtendedCPU[j];
			if (node1 >= node2) {
				continue;
			} else {
				result.push_back(std::make_pair<uint, uint>(node1, node2));
			}
			iterationCounter++;
		}
	}
	return result;
}

std::vector<std::pair<uint, uint> > SceNodes::obtainPossibleNeighborPairs_M() {
	std::vector<std::pair<uint, uint> > result;
	thrust::host_vector<uint> keyBeginCPU = auxVecs.keyBegin;
	thrust::host_vector<uint> keyEndCPU = auxVecs.keyEnd;
	thrust::host_vector<uint> bucketKeysCPU = auxVecs.bucketKeys;
	thrust::host_vector<uint> bucketValuesCPU = auxVecs.bucketValues;
	thrust::host_vector<uint> bucketValuesExtendedCPU =
			auxVecs.bucketValuesIncludingNeighbor;
	uint iterationCounter = 0;

	uint maxNodePerCell = allocPara_M.maxAllNodePerCell;
	uint offSet = allocPara_M.bdryNodeCount;
	uint memThreshold = allocPara_M.maxMembrNodePerCell;
	int size = bucketKeysCPU.size();

	int node1, node2, cellRank1, cellRank2, nodeRank1, nodeRank2;
	for (int i = 0; i < size; i++) {
		for (int j = keyBeginCPU[bucketKeysCPU[i]];
				j < keyEndCPU[bucketKeysCPU[i]]; j++) {
			node1 = bucketValuesCPU[i];
			node2 = bucketValuesExtendedCPU[j];
			if (node1 >= node2) {
				continue;
			} else {
				cellRank1 = (node1 - offSet) / maxNodePerCell;
				nodeRank1 = (node1 - offSet) % maxNodePerCell;
				cellRank2 = (node2 - offSet) / maxNodePerCell;
				nodeRank2 = (node2 - offSet) % maxNodePerCell;
				if (nodeRank1 >= memThreshold && nodeRank2 >= memThreshold
						&& cellRank1 == cellRank2) {
					result.push_back(std::make_pair<uint, uint>(node1, node2));
				}
			}
			iterationCounter++;
		}
	}
	return result;
}

void SceNodes::initValues(std::vector<CVector>& initBdryCellNodePos,
		std::vector<CVector>& initProfileNodePos,
		std::vector<CVector>& initCartNodePos,
		std::vector<CVector>& initECMNodePos,
		std::vector<CVector>& initFNMCellNodePos,
		std::vector<CVector>& initMXCellNodePos) {

	uint FNMNodeCount = initFNMCellNodePos.size();
	uint MXNodeCount = initMXCellNodePos.size();

	uint beginAddressOfProfile = allocPara.startPosProfile;
	uint beginAddressOfCart = allocPara.startPosCart;
// find the begining position of ECM.
	uint beginAddressOfECM = allocPara.startPosECM;
// find the begining position of FNM cells.
	uint beginAddressOfFNM = allocPara.startPosCells;
// find the begining position of MX cells.
	uint beginAddressOfMX = beginAddressOfFNM + FNMNodeCount;

	std::vector<double> initBdryCellNodePosX = getArrayXComp(
			initBdryCellNodePos);
	thrust::copy(initBdryCellNodePosX.begin(), initBdryCellNodePosX.end(),
			infoVecs.nodeLocX.begin());
	std::vector<double> initBdryCellNodePosY = getArrayYComp(
			initBdryCellNodePos);
	thrust::copy(initBdryCellNodePosY.begin(), initBdryCellNodePosY.end(),
			infoVecs.nodeLocY.begin());

// copy x and y position of nodes of Profile to actual node position.
	std::vector<double> initProfileNodePosX = getArrayXComp(initProfileNodePos);
	thrust::copy(initProfileNodePosX.begin(), initProfileNodePosX.end(),
			infoVecs.nodeLocX.begin() + beginAddressOfProfile);
	std::vector<double> initProfileNodePosY = getArrayYComp(initProfileNodePos);
	thrust::copy(initProfileNodePosY.begin(), initProfileNodePosY.end(),
			infoVecs.nodeLocY.begin() + beginAddressOfProfile);

// copy x and y position of nodes of Profile to actual node position.
	std::vector<double> initCartNodePosX = getArrayXComp(initCartNodePos);
	thrust::copy(initCartNodePosX.begin(), initCartNodePosX.end(),
			infoVecs.nodeLocX.begin() + beginAddressOfCart);
	std::vector<double> initCartNodePosY = getArrayYComp(initCartNodePos);
	thrust::copy(initCartNodePosY.begin(), initCartNodePosY.end(),
			infoVecs.nodeLocY.begin() + beginAddressOfCart);

// copy x and y position of nodes of ECM to actual node position.
	std::vector<double> initECMNodePosX = getArrayXComp(initECMNodePos);
	thrust::copy(initECMNodePosX.begin(), initECMNodePosX.end(),
			infoVecs.nodeLocX.begin() + beginAddressOfECM);
	std::vector<double> initECMNodePosY = getArrayYComp(initECMNodePos);
	thrust::copy(initECMNodePosY.begin(), initECMNodePosY.end(),
			infoVecs.nodeLocY.begin() + beginAddressOfECM);

	for (int i = 0; i < initECMNodePosX.size(); i++) {
		assert(infoVecs.nodeLocX[i + beginAddressOfECM] == initECMNodePosX[i]);
		assert(!isnan(initECMNodePosX[i]));
	}

// copy x and y position of nodes of FNM cells to actual node position.
	std::vector<double> initFNMCellNodePosX = getArrayXComp(initFNMCellNodePos);
	thrust::copy(initFNMCellNodePosX.begin(), initFNMCellNodePosX.end(),
			infoVecs.nodeLocX.begin() + beginAddressOfFNM);
	std::vector<double> initFNMCellNodePosY = getArrayYComp(initFNMCellNodePos);
	thrust::copy(initFNMCellNodePosY.begin(), initFNMCellNodePosY.end(),
			infoVecs.nodeLocY.begin() + beginAddressOfFNM);

	thrust::fill(infoVecs.nodeCellType.begin() + beginAddressOfFNM,
			infoVecs.nodeCellType.begin() + beginAddressOfMX, FNM);

// copy x and y position of nodes of MX cells to actual node position.
	std::vector<double> initMXCellNodePosX = getArrayXComp(initMXCellNodePos);
	thrust::copy(initMXCellNodePosX.begin(), initMXCellNodePosX.end(),
			infoVecs.nodeLocX.begin() + beginAddressOfMX);
	std::vector<double> initMXCellNodePosY = getArrayYComp(initMXCellNodePos);
	thrust::copy(initMXCellNodePosY.begin(), initMXCellNodePosY.end(),
			infoVecs.nodeLocY.begin() + beginAddressOfMX);

	thrust::fill(infoVecs.nodeCellType.begin() + beginAddressOfMX,
			infoVecs.nodeCellType.begin() + beginAddressOfMX + MXNodeCount, MX);
}

void SceNodes::initValues_M(std::vector<bool>& initIsActive,
		std::vector<CVector>& initCellNodePos,
		std::vector<SceNodeType>& nodeTypes) {

	std::vector<double> initCellNodePosX = getArrayXComp(initCellNodePos);
	thrust::copy(initCellNodePosX.begin(), initCellNodePosX.end(),
			infoVecs.nodeLocX.begin() + allocPara_M.bdryNodeCount);
	std::vector<double> initCellNodePosY = getArrayYComp(initCellNodePos);
	thrust::copy(initCellNodePosY.begin(), initCellNodePosY.end(),
			infoVecs.nodeLocY.begin() + allocPara_M.bdryNodeCount);

	thrust::copy(nodeTypes.begin(), nodeTypes.end(),
			infoVecs.nodeCellType.begin() + allocPara_M.bdryNodeCount);

	thrust::copy(initIsActive.begin(), initIsActive.end(),
			infoVecs.nodeIsActive.begin() + allocPara_M.bdryNodeCount);

}

void SceNodes::applyProfileForces() {
	thrust::counting_iterator<uint> countingIterBegin(0);
	thrust::counting_iterator<uint> countingIterEnd(
			allocPara.currentActiveProfileNodeCount);

	double* nodeLocXAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeLocX[allocPara.startPosProfile]);
	double* nodeLocYAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeLocY[allocPara.startPosProfile]);
	double* nodeLocZAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeLocZ[allocPara.startPosProfile]);

	double* nodeVelXAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeVelX[allocPara.startPosProfile]);
	double* nodeVelYAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeVelY[allocPara.startPosProfile]);
	double* nodeVelZAddressEpiBegin = thrust::raw_pointer_cast(
			&infoVecs.nodeVelZ[allocPara.startPosProfile]);

	thrust::transform(countingIterBegin, countingIterEnd,
			thrust::make_zip_iterator(
					thrust::make_tuple(infoVecs.nodeVelX.begin(),
							infoVecs.nodeVelY.begin(),
							infoVecs.nodeVelZ.begin()))
					+ allocPara.startPosProfile,
			AddLinkForces(nodeLocXAddressEpiBegin, nodeLocYAddressEpiBegin,
					nodeLocZAddressEpiBegin, nodeVelXAddressEpiBegin,
					nodeVelYAddressEpiBegin, nodeVelZAddressEpiBegin,
					allocPara.currentActiveProfileNodeCount));
}

VtkAnimationData SceNodes::obtainAnimationData(AnimationCriteria aniCri) {
	VtkAnimationData vtkData;
	std::vector<std::pair<uint, uint> > pairs = obtainPossibleNeighborPairs();
	cout << "size of potential pairs = " << pairs.size() << endl;

// unordered_map is more efficient than map, but it is a c++ 11 feature
// and c++ 11 seems to be incompatible with Thrust.
	IndexMap locIndexToAniIndexMap;

// Doesn't have to copy the entire nodeLocX array.
// Only copy the first half will be sufficient
	thrust::host_vector<double> hostTmpVectorLocX = infoVecs.nodeLocX;
	thrust::host_vector<double> hostTmpVectorLocY = infoVecs.nodeLocY;
	thrust::host_vector<double> hostTmpVectorLocZ = infoVecs.nodeLocZ;

	thrust::host_vector<double> hostTmpVectorForceX;
	thrust::host_vector<double> hostTmpVectorForceY;
	thrust::host_vector<double> hostTmpVectorForceZ;
	thrust::host_vector<double> hostTmpVectorVelVal;

	assert(hostTmpVectorLocX.size() == hostTmpVectorLocY.size());
	assert(hostTmpVectorLocY.size() == hostTmpVectorLocZ.size());

	thrust::host_vector<SceNodeType> hostTmpVectorNodeType =
			infoVecs.nodeCellType;
	thrust::host_vector<uint> hostTmpVectorNodeRank = infoVecs.nodeCellRank;
	thrust::host_vector<double> hostTmpVectorNodeStress;

	if (aniCri.animationType != CellType) {
		hostTmpVectorForceX = infoVecs.nodeInterForceX;
		hostTmpVectorForceY = infoVecs.nodeInterForceY;
		hostTmpVectorForceZ = infoVecs.nodeInterForceZ;

		assert(hostTmpVectorForceX.size() == hostTmpVectorLocX.size());
		assert(hostTmpVectorForceX.size() == hostTmpVectorForceY.size());
		assert(hostTmpVectorForceX.size() == hostTmpVectorForceZ.size());

		uint vecSize = hostTmpVectorForceX.size();
		hostTmpVectorVelVal.resize(vecSize);
		for (uint i = 0; i < vecSize; i++) {
			hostTmpVectorVelVal[i] = sqrt(
					hostTmpVectorForceX[i] * hostTmpVectorForceX[i]
							+ hostTmpVectorForceY[i] * hostTmpVectorForceY[i]
							+ hostTmpVectorForceZ[i] * hostTmpVectorForceZ[i]);
		}

	}
	if (aniCri.animationType == Force) {
		vtkData.isArrowIncluded = true;
	} else {
		vtkData.isArrowIncluded = false;
	}

	uint curIndex = 0;
	for (uint i = 0; i < pairs.size(); i++) {
		uint node1Index = pairs[i].first;
		uint node2Index = pairs[i].second;
		double node1X = hostTmpVectorLocX[node1Index];
		double node1Y = hostTmpVectorLocY[node1Index];
		double node1Z = hostTmpVectorLocZ[node1Index];
		SceNodeType node1T = hostTmpVectorNodeType[node1Index];
		uint node1R = hostTmpVectorNodeRank[node1Index];
		double node2X = hostTmpVectorLocX[node2Index];
		double node2Y = hostTmpVectorLocY[node2Index];
		double node2Z = hostTmpVectorLocZ[node2Index];
		SceNodeType node2T = hostTmpVectorNodeType[node2Index];
		uint node2R = hostTmpVectorNodeRank[node2Index];

		if (aniCri.isPairQualify(node1Index, node2Index, node1X, node1Y, node1Z,
				node1T, node1R, node2X, node2Y, node2Z, node2T, node2R)) {
			IndexMap::iterator it = locIndexToAniIndexMap.find(pairs[i].first);
			if (it == locIndexToAniIndexMap.end()) {
				locIndexToAniIndexMap.insert(
						std::pair<uint, uint>(pairs[i].first, curIndex));
				curIndex++;
				PointAniData ptAniData;
				if (aniCri.animationType == ForceAbsVal) {
					ptAniData.colorScale = hostTmpVectorVelVal[node1Index];
				} else if (aniCri.animationType == Force) {
					ptAniData.colorScale = hostTmpVectorVelVal[node1Index];
					if (hostTmpVectorVelVal[node1Index] > aniCri.threshold) {
						ptAniData.dir.x = hostTmpVectorForceX[node1Index]
								/ hostTmpVectorVelVal[node1Index]
								* aniCri.arrowLength;
						ptAniData.dir.y = hostTmpVectorForceY[node1Index]
								/ hostTmpVectorVelVal[node1Index]
								* aniCri.arrowLength;
						ptAniData.dir.z = hostTmpVectorForceZ[node1Index]
								/ hostTmpVectorVelVal[node1Index]
								* aniCri.arrowLength;
					} else {
						ptAniData.dir.x = 0;
						ptAniData.dir.y = 0;
						ptAniData.dir.z = 0;
					}
				} else {
					ptAniData.colorScale = nodeTypeToScale(node1T);
				}
				ptAniData.pos = CVector(node1X, node1Y, node1Z);
				vtkData.pointsAniData.push_back(ptAniData);
			}
			it = locIndexToAniIndexMap.find(pairs[i].second);
			if (it == locIndexToAniIndexMap.end()) {
				locIndexToAniIndexMap.insert(
						std::pair<uint, uint>(pairs[i].second, curIndex));
				curIndex++;
				PointAniData ptAniData;
				if (aniCri.animationType == ForceAbsVal) {
					ptAniData.colorScale = hostTmpVectorVelVal[node2Index];
				} else if (aniCri.animationType == Force) {
					ptAniData.colorScale = hostTmpVectorVelVal[node2Index];
					if (hostTmpVectorVelVal[node2Index] > aniCri.threshold) {
						ptAniData.dir.x = hostTmpVectorForceX[node2Index]
								/ hostTmpVectorVelVal[node2Index]
								* aniCri.arrowLength;
						ptAniData.dir.y = hostTmpVectorForceY[node2Index]
								/ hostTmpVectorVelVal[node2Index]
								* aniCri.arrowLength;
						ptAniData.dir.z = hostTmpVectorForceZ[node2Index]
								/ hostTmpVectorVelVal[node2Index]
								* aniCri.arrowLength;
					} else {
						ptAniData.dir.x = 0;
						ptAniData.dir.y = 0;
						ptAniData.dir.z = 0;
					}
				} else {
					ptAniData.colorScale = nodeTypeToScale(node2T);
				}
				ptAniData.pos = CVector(node2X, node2Y, node2Z);
				vtkData.pointsAniData.push_back(ptAniData);
			}

			it = locIndexToAniIndexMap.find(pairs[i].first);
			uint aniIndex1 = it->second;
			it = locIndexToAniIndexMap.find(pairs[i].second);
			uint aniIndex2 = it->second;

			LinkAniData linkData;
			linkData.node1Index = aniIndex1;
			linkData.node2Index = aniIndex2;
			vtkData.linksAniData.push_back(linkData);
		}
	}

	uint profileStartIndex = allocPara.startPosProfile;
	uint profileEndIndex = profileStartIndex
			+ allocPara.currentActiveProfileNodeCount;

	for (uint i = profileStartIndex; i < profileEndIndex; i++) {
		PointAniData ptAniData;
		ptAniData.pos = CVector(hostTmpVectorLocX[i], hostTmpVectorLocY[i],
				hostTmpVectorLocZ[i]);

		if (aniCri.animationType == ForceAbsVal) {
			ptAniData.colorScale = hostTmpVectorVelVal[i];
		} else if (aniCri.animationType == Force) {
			ptAniData.colorScale = hostTmpVectorVelVal[i];
			if (hostTmpVectorVelVal[i] > aniCri.threshold) {
				ptAniData.dir.x = hostTmpVectorForceX[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
				ptAniData.dir.y = hostTmpVectorForceY[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
				ptAniData.dir.z = hostTmpVectorForceZ[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
			}
		} else {
			ptAniData.colorScale = nodeTypeToScale(hostTmpVectorNodeType[i]);
		}
		vtkData.pointsAniData.push_back(ptAniData);
		LinkAniData linkData;
		linkData.node1Index = curIndex;
		linkData.node2Index = curIndex + 1;
		if (i != profileEndIndex - 1) {
			vtkData.linksAniData.push_back(linkData);
		}
		curIndex++;
	}

	uint cartStartIndex = allocPara.startPosCart;
	uint cartEndIndex = cartStartIndex + allocPara.maxCartNodeCount;
	for (uint i = cartStartIndex; i < cartEndIndex; i++) {
		bool isActive = infoVecs.nodeIsActive[i];
		if (!isActive) {
			continue;
		}

		PointAniData ptAniData;
		ptAniData.pos = CVector(hostTmpVectorLocX[i], hostTmpVectorLocY[i],
				hostTmpVectorLocZ[i]);
		if (aniCri.animationType == ForceAbsVal) {
			ptAniData.colorScale = hostTmpVectorVelVal[i];
		} else if (aniCri.animationType == Force) {
			ptAniData.colorScale = hostTmpVectorVelVal[i];
			if (hostTmpVectorVelVal[i] > aniCri.threshold) {
				ptAniData.dir.x = hostTmpVectorForceX[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
				ptAniData.dir.y = hostTmpVectorForceY[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
				ptAniData.dir.z = hostTmpVectorForceZ[i]
						/ hostTmpVectorVelVal[i] * aniCri.arrowLength;
			}
		} else {
			ptAniData.colorScale = nodeTypeToScale(hostTmpVectorNodeType[i]);
		}
		vtkData.pointsAniData.push_back(ptAniData);

		bool isNextActive;
		if (i == cartEndIndex - 1) {
			isNextActive = false;
		} else {
			isNextActive = infoVecs.nodeIsActive[i + 1];
		}
		if (isNextActive) {
			LinkAniData linkData;
			linkData.node1Index = curIndex;
			linkData.node2Index = curIndex + 1;
			vtkData.linksAniData.push_back(linkData);
		}
		curIndex++;
	}

	return vtkData;
}

// TODO
VtkAnimationData SceNodes::obtainAnimationData_M(AnimationCriteria aniCri) {
	VtkAnimationData vtkData;
	std::vector<std::pair<uint, uint> > pairs = obtainPossibleNeighborPairs_M();
	cout << "size of potential pairs = " << pairs.size() << endl;

	// unordered_map is more efficient than map, but it is a c++ 11 feature
	// and c++ 11 seems to be incompatible with Thrust.
	IndexMap locIndexToAniIndexMap;

	// Doesn't have to copy the entire nodeLocX array.
	// Only copy the first half will be sufficient
	thrust::host_vector<double> hostTmpVectorLocX = infoVecs.nodeLocX;
	thrust::host_vector<double> hostTmpVectorLocY = infoVecs.nodeLocY;
	thrust::host_vector<bool> hostIsActiveVec = infoVecs.nodeIsActive;
	thrust::host_vector<int> hostBondVec = infoVecs.nodeAdhereIndex;

	uint activeCellCount = allocPara_M.currentActiveCellCount;
	uint maxNodePerCell = allocPara_M.maxAllNodePerCell;
	uint maxMemNodePerCell = allocPara_M.maxMembrNodePerCell;
	uint beginIndx = allocPara_M.bdryNodeCount;
	//uint endIndx = beginIndx + activeCellCount * maxNodePerCell;

	//uint cellRank1, nodeRank1, cellRank2, nodeRank2;
	uint index1;
	int index2;
	std::vector<BondInfo> bondInfoVec;

	for (uint i = 0; i < activeCellCount; i++) {
		for (uint j = 0; j < maxMemNodePerCell; j++) {
			index1 = beginIndx + i * maxNodePerCell + j;
			if (hostIsActiveVec[index1] == true) {
				index2 = hostBondVec[index1];
				if (index2 > index1 && index2 != -1) {
					BondInfo bond;
					bond.cellRank1 = i;
					bond.pos1 = CVector(hostTmpVectorLocX[index1],
							hostTmpVectorLocY[index1], 0);
					bond.cellRank2 = (index2 - beginIndx) / maxNodePerCell;
					bond.pos2 = CVector(hostTmpVectorLocX[index2],
							hostTmpVectorLocY[index2], 0);
					bondInfoVec.push_back(bond);
				}
			}
		}
	}
	vtkData.bondsInfo = bondInfoVec;

	thrust::host_vector<SceNodeType> hostTmpVectorNodeType =
			infoVecs.nodeCellType;

	uint curIndex = 0;
	for (uint i = 0; i < pairs.size(); i++) {
		uint node1Index = pairs[i].first;
		uint node2Index = pairs[i].second;
		double node1X = hostTmpVectorLocX[node1Index];
		double node1Y = hostTmpVectorLocY[node1Index];

		double node2X = hostTmpVectorLocX[node2Index];
		double node2Y = hostTmpVectorLocY[node2Index];

		if (aniCri.isPairQualify_M(node1X, node1Y, node2X, node2Y)) {
			IndexMap::iterator it = locIndexToAniIndexMap.find(pairs[i].first);
			if (it == locIndexToAniIndexMap.end()) {
				locIndexToAniIndexMap.insert(
						std::pair<uint, uint>(pairs[i].first, curIndex));
				curIndex++;
				PointAniData ptAniData;
				ptAniData.colorScale = nodeTypeToScale(
						hostTmpVectorNodeType[node1Index]);
				ptAniData.pos = CVector(node1X, node1Y, 0);
				vtkData.pointsAniData.push_back(ptAniData);
			}
			it = locIndexToAniIndexMap.find(pairs[i].second);
			if (it == locIndexToAniIndexMap.end()) {
				locIndexToAniIndexMap.insert(
						std::pair<uint, uint>(pairs[i].second, curIndex));
				curIndex++;
				PointAniData ptAniData;
				ptAniData.colorScale = nodeTypeToScale(
						hostTmpVectorNodeType[node1Index]);
				ptAniData.pos = CVector(node2X, node2Y, 0);
				vtkData.pointsAniData.push_back(ptAniData);
			}

			it = locIndexToAniIndexMap.find(pairs[i].first);
			uint aniIndex1 = it->second;
			it = locIndexToAniIndexMap.find(pairs[i].second);
			uint aniIndex2 = it->second;

			LinkAniData linkData;
			linkData.node1Index = aniIndex1;
			linkData.node2Index = aniIndex2;
			vtkData.linksAniData.push_back(linkData);
		}
	}

	return vtkData;
}

void SceNodes::findBucketBounds() {
	thrust::counting_iterator<unsigned int> search_begin(0);
	thrust::lower_bound(auxVecs.bucketKeysExpanded.begin(),
			auxVecs.bucketKeysExpanded.end(), search_begin,
			search_begin + domainPara.totalBucketCount,
			auxVecs.keyBegin.begin());
	thrust::upper_bound(auxVecs.bucketKeysExpanded.begin(),
			auxVecs.bucketKeysExpanded.end(), search_begin,
			search_begin + domainPara.totalBucketCount, auxVecs.keyEnd.begin());
}

void SceNodes::prepareSceForceComputation() {
	buildBuckets2D();
	extendBuckets2D();
	findBucketBounds();
}

void SceNodes::addNewlyDividedCells(
		thrust::device_vector<double> &nodeLocXNewCell,
		thrust::device_vector<double> &nodeLocYNewCell,
		thrust::device_vector<double> &nodeLocZNewCell,
		thrust::device_vector<bool> &nodeIsActiveNewCell,
		thrust::device_vector<SceNodeType> &nodeCellTypeNewCell) {

// data validation
	uint nodesSize = nodeLocXNewCell.size();
	assert(nodesSize % allocPara.maxNodeOfOneCell == 0);
	uint addCellCount = nodesSize / allocPara.maxNodeOfOneCell;

// position that we will add newly divided cells.
	uint shiftStartPosNewCell = allocPara.startPosCells
			+ allocPara.currentActiveCellCount * allocPara.maxNodeOfOneCell;

	thrust::copy(
			thrust::make_zip_iterator(
					thrust::make_tuple(nodeLocXNewCell.begin(),
							nodeLocYNewCell.begin(), nodeLocZNewCell.begin(),
							nodeIsActiveNewCell.begin(),
							nodeCellTypeNewCell.begin())),
			thrust::make_zip_iterator(
					thrust::make_tuple(nodeLocXNewCell.end(),
							nodeLocYNewCell.end(), nodeLocZNewCell.end(),
							nodeIsActiveNewCell.end(),
							nodeCellTypeNewCell.end())),
			thrust::make_zip_iterator(
					thrust::make_tuple(infoVecs.nodeLocX.begin(),
							infoVecs.nodeLocY.begin(),
							infoVecs.nodeLocZ.begin(),
							infoVecs.nodeIsActive.begin(),
							infoVecs.nodeCellType.begin()))
					+ shiftStartPosNewCell);

// total number of cells has increased.
	allocPara.currentActiveCellCount = allocPara.currentActiveCellCount
			+ addCellCount;
}

void SceNodes::buildBuckets2D() {
	int totalActiveNodes;
	if (controlPara.simuType != Disc_M) {
		totalActiveNodes = allocPara.startPosCells
				+ allocPara.currentActiveCellCount * allocPara.maxNodeOfOneCell;
	} else {
		totalActiveNodes = allocPara_M.bdryNodeCount
				+ allocPara_M.currentActiveCellCount
						* allocPara_M.maxAllNodePerCell;
	}

	auxVecs.bucketKeys.resize(totalActiveNodes);
	auxVecs.bucketValues.resize(totalActiveNodes);
	thrust::counting_iterator<uint> countingIterBegin(0);
	thrust::counting_iterator<uint> countingIterEnd(totalActiveNodes);

// takes counting iterator and coordinates
// return tuple of keys and values
// transform the points to their bucket indices
	thrust::transform(
			make_zip_iterator(
					make_tuple(infoVecs.nodeLocX.begin(),
							infoVecs.nodeLocY.begin(),
							infoVecs.nodeLocZ.begin(),
							infoVecs.nodeIsActive.begin(), countingIterBegin)),
			make_zip_iterator(
					make_tuple(infoVecs.nodeLocX.begin(),
							infoVecs.nodeLocY.begin(),
							infoVecs.nodeLocZ.begin(),
							infoVecs.nodeIsActive.begin(), countingIterBegin))
					+ totalActiveNodes,
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeys.begin(),
							auxVecs.bucketValues.begin())),
			pointToBucketIndex2D(domainPara.minX, domainPara.maxX,
					domainPara.minY, domainPara.maxY, domainPara.gridSpacing));

// sort the points by their bucket index
	thrust::sort_by_key(auxVecs.bucketKeys.begin(), auxVecs.bucketKeys.end(),
			auxVecs.bucketValues.begin());
// for those nodes that are inactive, key value of UINT_MAX will be returned.
// we need to removed those keys along with their values.
	int numberOfOutOfRange = thrust::count(auxVecs.bucketKeys.begin(),
			auxVecs.bucketKeys.end(), UINT_MAX);

	auxVecs.bucketKeys.erase(auxVecs.bucketKeys.end() - numberOfOutOfRange,
			auxVecs.bucketKeys.end());
	auxVecs.bucketValues.erase(auxVecs.bucketValues.end() - numberOfOutOfRange,
			auxVecs.bucketValues.end());
}
__device__
double computeDist(double &xPos, double &yPos, double &zPos, double &xPos2,
		double &yPos2, double &zPos2) {
	return sqrt(
			(xPos - xPos2) * (xPos - xPos2) + (yPos - yPos2) * (yPos - yPos2)
					+ (zPos - zPos2) * (zPos - zPos2));
}

__device__
double computeDist2D(double &xPos, double &yPos, double &xPos2, double &yPos2) {
	return sqrt(
			(xPos - xPos2) * (xPos - xPos2) + (yPos - yPos2) * (yPos - yPos2));
}

__device__
void calculateAndAddECMForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {

	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceECMPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceECMPara[0] / sceECMPara[2]
				* exp(-linkLength / sceECMPara[2])
				+ sceECMPara[1] / sceECMPara[3]
						* exp(-linkLength / sceECMPara[3]);
		if (forceValue > 0) {
			//forceValue = 0;
			forceValue = forceValue * 0.3;
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}
__device__
void calculateAndAddProfileForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	forceValue = -sceProfilePara[5] * (linkLength - sceProfilePara[6]);

	if (linkLength > 1.0e-12) {
		xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
		yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
		zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
	}
}

__device__
void calculateAndAddIntraForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue;
	if (linkLength > sceIntraPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceIntraPara[0] / sceIntraPara[2]
				* exp(-linkLength / sceIntraPara[2])
				+ sceIntraPara[1] / sceIntraPara[3]
						* exp(-linkLength / sceIntraPara[3]);
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}

__device__
void calAndAddIntraForceDiv(double& xPos, double& yPos, double& zPos,
		double& xPos2, double& yPos2, double& zPos2, double& growPro,
		double& xRes, double& yRes, double& zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue;
	if (linkLength > sceIntraPara[4]) {
		forceValue = 0;
	} else {
		if (growPro > sceIntraParaDiv[4]) {
			double intraPara0 = growPro * (sceIntraParaDiv[0])
					+ (1.0 - growPro) * sceIntraPara[0];
			double intraPara1 = growPro * (sceIntraParaDiv[1])
					+ (1.0 - growPro) * sceIntraPara[1];
			double intraPara2 = growPro * (sceIntraParaDiv[2])
					+ (1.0 - growPro) * sceIntraPara[2];
			double intraPara3 = growPro * (sceIntraParaDiv[3])
					+ (1.0 - growPro) * sceIntraPara[3];
			forceValue = -intraPara0 / intraPara2
					* exp(-linkLength / intraPara2)
					+ intraPara1 / intraPara3 * exp(-linkLength / intraPara3);
		} else {
			forceValue = -sceIntraPara[0] / sceIntraPara[2]
					* exp(-linkLength / sceIntraPara[2])
					+ sceIntraPara[1] / sceIntraPara[3]
							* exp(-linkLength / sceIntraPara[3]);
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}

__device__
void calAndAddIntraDiv_M(double& xPos, double& yPos, double& xPos2,
		double& yPos2, double& growPro, double& xRes, double& yRes) {
	double linkLength = computeDist2D(xPos, yPos, xPos2, yPos2);
	double forceValue;
	if (linkLength > sceIntraPara[4]) {
		forceValue = 0;
	} else {
		if (growPro > sceIntraParaDiv[4]) {
			double intraPara0 = growPro * (sceIntraParaDiv[0])
					+ (1.0 - growPro) * sceIntraPara[0];
			double intraPara1 = growPro * (sceIntraParaDiv[1])
					+ (1.0 - growPro) * sceIntraPara[1];
			double intraPara2 = growPro * (sceIntraParaDiv[2])
					+ (1.0 - growPro) * sceIntraPara[2];
			double intraPara3 = growPro * (sceIntraParaDiv[3])
					+ (1.0 - growPro) * sceIntraPara[3];
			forceValue = -intraPara0 / intraPara2
					* exp(-linkLength / intraPara2)
					+ intraPara1 / intraPara3 * exp(-linkLength / intraPara3);
		} else {
			forceValue = -sceIntraPara[0] / sceIntraPara[2]
					* exp(-linkLength / sceIntraPara[2])
					+ sceIntraPara[1] / sceIntraPara[3]
							* exp(-linkLength / sceIntraPara[3]);
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
}

__device__
void calAndAddIntraB_M(double& xPos, double& yPos, double& xPos2, double& yPos2,
		double& xRes, double& yRes) {
	double linkLength = computeDist2D(xPos, yPos, xPos2, yPos2);
	double forceValue;
	if (linkLength > sceIntraBPara_M[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceIntraBPara_M[0] / sceIntraBPara_M[2]
				* exp(-linkLength / sceIntraBPara_M[2])
				+ sceIntraBPara_M[1] / sceIntraBPara_M[3]
						* exp(-linkLength / sceIntraBPara_M[3]);
	}
	if (forceValue > 0) {
		forceValue = 0;
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
}

__device__
void calAndAddInter_M(double& xPos, double& yPos, double& xPos2, double& yPos2,
		double& xRes, double& yRes) {
	double linkLength = computeDist2D(xPos, yPos, xPos2, yPos2);
	double forceValue;
	if (linkLength > sceInterPara_M[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceInterPara_M[0] / sceInterPara_M[2]
				* exp(-linkLength / sceInterPara_M[2])
				+ sceInterPara_M[1] / sceInterPara_M[3]
						* exp(-linkLength / sceInterPara_M[3]);
		if (forceValue > 0) {
			forceValue = 0;
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
}

__device__
void calculateAndAddInterForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceInterPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceInterPara[0] / sceInterPara[2]
				* exp(-linkLength / sceInterPara[2])
				+ sceInterPara[1] / sceInterPara[3]
						* exp(-linkLength / sceInterPara[3]);
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}

__device__
void calAndAddInterForceDisc(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes, double& interForceX, double& interForceY,
		double& interForceZ) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceInterPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceInterPara[0] / sceInterPara[2]
				* exp(-linkLength / sceInterPara[2])
				+ sceInterPara[1] / sceInterPara[3]
						* exp(-linkLength / sceInterPara[3]);
	}
	double fX = forceValue * (xPos2 - xPos) / linkLength;
	double fY = forceValue * (yPos2 - yPos) / linkLength;
	double fZ = forceValue * (zPos2 - zPos) / linkLength;
	xRes = xRes + fX;
	yRes = yRes + fY;
	zRes = zRes + fZ;
	interForceX = interForceX + fX;
	interForceY = interForceY + fY;
	interForceZ = interForceZ + fZ;
}

__device__
void calculateAndAddCartForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceCartPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceCartPara[0] / sceCartPara[2]
				* exp(-linkLength / sceCartPara[2])
				+ sceCartPara[1] / sceCartPara[3]
						* exp(-linkLength / sceCartPara[3]);
		if (linkLength > 1.0e-12) {
			//double dotProduct = (xPos2 - xPos) / linkLength * cartGrowDirVec[0]
			//		+ (yPos2 - yPos) / linkLength * cartGrowDirVec[1]
			//		+ (zPos2 - zPos) / linkLength * cartGrowDirVec[2];
			//forceValue = forceValue * dotProduct;
			// this is just a temperary solution -- the direction should not be fixed.
			xRes = xRes - forceValue * cartGrowDirVec[0];
			yRes = yRes - forceValue * cartGrowDirVec[1];
			zRes = zRes - forceValue * cartGrowDirVec[2];
			//xRes = xRes + forceValue * (xPos2 - xPos);
			//yRes = yRes + forceValue * (yPos2 - yPos);
			//zRes = zRes + forceValue * (zPos2 - zPos);
		}
		if (forceValue > 0) {
			//forceValue = forceValue * 0.01;
			forceValue = 0;
			//xRes = xRes + forceValue * (xPos2 - xPos);
			//yRes = yRes + forceValue * (yPos2 - yPos);
			//zRes = zRes + forceValue * (zPos2 - zPos);
		}
	}

}

__device__
void calculateAndAddDiffInterCellForce(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceInterDiffPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceInterDiffPara[0] / sceInterDiffPara[2]
				* exp(-linkLength / sceInterDiffPara[2])
				+ sceInterDiffPara[1] / sceInterDiffPara[3]
						* exp(-linkLength / sceInterDiffPara[3]);
		if (forceValue > 0) {
			//forceValue = 0;
			forceValue = forceValue * 0.2;
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}
__device__
void calculateAndAddInterForceDiffType(double &xPos, double &yPos, double &zPos,
		double &xPos2, double &yPos2, double &zPos2, double &xRes, double &yRes,
		double &zRes) {
	double linkLength = computeDist(xPos, yPos, zPos, xPos2, yPos2, zPos2);
	double forceValue = 0;
	if (linkLength > sceInterPara[4]) {
		forceValue = 0;
	} else {
		forceValue = -sceInterPara[0] / sceInterPara[2]
				* exp(-linkLength / sceInterPara[2])
				+ sceInterPara[1] / sceInterPara[3]
						* exp(-linkLength / sceInterPara[3]);
		if (forceValue > 0) {
			//forceValue = 0;
			forceValue = forceValue * 0.3;
		}
	}
	xRes = xRes + forceValue * (xPos2 - xPos) / linkLength;
	yRes = yRes + forceValue * (yPos2 - yPos) / linkLength;
	zRes = zRes + forceValue * (zPos2 - zPos) / linkLength;
}

__device__ bool bothNodesCellNode(uint nodeGlobalRank1, uint nodeGlobalRank2,
		uint cellNodesThreshold) {
	if (nodeGlobalRank1 < cellNodesThreshold
			&& nodeGlobalRank2 < cellNodesThreshold) {
		return true;
	} else {
		return false;
	}
}

__device__ bool isSameCell(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if (nodeGlobalRank1 < cellNodeBeginPos
			|| nodeGlobalRank2 < cellNodeBeginPos) {
		return false;
	}
	if ((nodeGlobalRank1 - cellNodeBeginPos) / nodeCountPerCell
			== (nodeGlobalRank2 - cellNodeBeginPos) / nodeCountPerCell) {
		return true;
	} else {
		return false;
	}
}

__device__
bool isSameCell_m(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if (nodeGlobalRank1 < cellNodeBeginPos_M
			|| nodeGlobalRank2 < cellNodeBeginPos_M) {
		return false;
	}
	if ((nodeGlobalRank1 - cellNodeBeginPos_M) / allNodeCountPerCell_M
			== (nodeGlobalRank2 - cellNodeBeginPos_M) / allNodeCountPerCell_M) {
		return true;
	} else {
		return false;
	}
}

__device__
bool bothInternal(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if (nodeGlobalRank1 < cellNodeBeginPos_M
			|| nodeGlobalRank2 < cellNodeBeginPos_M) {
		return false;
	}
	uint nodeRank1 = (nodeGlobalRank1 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	uint nodeRank2 = (nodeGlobalRank2 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	if (nodeRank1 >= bdryThreshold_M && nodeRank2 >= bdryThreshold_M) {
		return true;
	} else {
		return false;
	}
}

__device__
bool bothEpi(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if (nodeGlobalRank1 < cellNodeBeginPos_M
			|| nodeGlobalRank2 < cellNodeBeginPos_M) {
		return false;
	}
	uint nodeRank1 = (nodeGlobalRank1 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	uint nodeRank2 = (nodeGlobalRank2 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	if (nodeRank1 < bdryThreshold_M && nodeRank2 < bdryThreshold_M) {
		return true;
	} else {
		return false;
	}
}

__device__
bool bothEpiDiffCell(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if (nodeGlobalRank1 < cellNodeBeginPos_M
			|| nodeGlobalRank2 < cellNodeBeginPos_M) {
		return false;
	}
	uint nodeRank1 = (nodeGlobalRank1 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	uint nodeRank2 = (nodeGlobalRank2 - cellNodeBeginPos_M)
			% allNodeCountPerCell_M;
	if (nodeRank1 == nodeRank2) {
		return false;
	}
	if (nodeRank1 < bdryThreshold_M && nodeRank2 < bdryThreshold_M) {
		return true;
	} else {
		return false;
	}
}

__device__ bool isSameECM(uint nodeGlobalRank1, uint nodeGlobalRank2) {
	if ((nodeGlobalRank1 - ECMbeginPos) / nodeCountPerECM
			== (nodeGlobalRank2 - ECMbeginPos) / nodeCountPerECM) {
		return true;
	} else {
		return false;
	}
}

__device__ bool isNeighborECMNodes(uint nodeGlobalRank1, uint nodeGlobalRank2) {
// this means that two nodes are from the same ECM
	if ((nodeGlobalRank1 - ECMbeginPos) / nodeCountPerECM
			== (nodeGlobalRank2 - ECMbeginPos) / nodeCountPerECM) {
		// this means that two nodes are actually close to each other
		// seems to be strange because of unsigned int.
		if ((nodeGlobalRank1 > nodeGlobalRank2
				&& nodeGlobalRank1 - nodeGlobalRank2 == 1)
				|| (nodeGlobalRank2 > nodeGlobalRank1
						&& nodeGlobalRank2 - nodeGlobalRank1 == 1)) {
			return true;
		}
	}
	return false;
}

__device__ bool isNeighborProfileNodes(uint nodeGlobalRank1,
		uint nodeGlobalRank2) {
	if ((nodeGlobalRank1 > nodeGlobalRank2
			&& nodeGlobalRank1 - nodeGlobalRank2 == 1)
			|| (nodeGlobalRank2 > nodeGlobalRank1
					&& nodeGlobalRank2 - nodeGlobalRank1 == 1)) {
		return true;
	}
	return false;
}

__device__ bool ofSameType(uint cellType1, uint cellType2) {
	if (cellType1 == cellType2) {
		return true;
	} else {
		return false;
	}
}

__device__ bool bothCellNodes(SceNodeType &type1, SceNodeType &type2) {
	if ((type1 == MX || type1 == FNM) && (type2 == MX || type2 == FNM)) {
		return true;
	} else {
		return false;
	}
}

__device__
void attemptToAdhere(bool& isSuccess, uint& index, double& dist,
		uint& nodeRank2, double& xPos1, double& yPos1, double& xPos2,
		double& yPos2) {
	double length = computeDist2D(xPos1, yPos1, xPos2, yPos2);
	if (length <= bondAdhThreshold) {
		if (isSuccess) {
			if (length < dist) {
				dist = length;
				index = nodeRank2;
			}
		} else {
			isSuccess = true;
			index = nodeRank2;
			dist = length;
		}
	}
}

__device__
void handleAdhesionForce_M(uint& nodeRank, int& adhereIndex, double& xPos,
		double& yPos, double* _nodeLocXAddress, double* _nodeLocYAddress,
		double& xRes, double& yRes) {

// should old one break?
	if (adhereIndex != -1) {
		// means adhesion has been established
		double curAdherePosX = _nodeLocXAddress[adhereIndex];
		double curAdherePosY = _nodeLocYAddress[adhereIndex];
		double curLen = computeDist2D(xPos, yPos, curAdherePosX, curAdherePosY);
		if (curLen > maxAdhBondLength) {
			adhereIndex = -1;
			return;
		} else {
			if (curLen > minAdhBondLength) {
				double forceValue = (curLen - minAdhBondLength) * bondStiff;
				xRes = xRes + forceValue * (curAdherePosX - xPos) / curLen;
				yRes = yRes + forceValue * (curAdherePosY - yPos) / curLen;
			}
		}
	}
}

__device__
void calculateForceBetweenLinkNodes(double &xLoc, double &yLoc, double &zLoc,
		double &xLocLeft, double &yLocLeft, double &zLocLeft, double &xLocRight,
		double &yLocRight, double &zLocRight, double &xVel, double &yVel,
		double &zVel) {
	double linkLengthLeft = computeDist(xLoc, yLoc, zLoc, xLocLeft, yLocLeft,
			zLocLeft);
	double forceValueLeft = sceProfilePara[5]
			* (linkLengthLeft - sceProfilePara[6]);
	xVel = xVel + forceValueLeft * (xLocLeft - xLoc) / linkLengthLeft;
	yVel = yVel + forceValueLeft * (yLocLeft - yLoc) / linkLengthLeft;
	zVel = zVel + forceValueLeft * (zLocLeft - zLoc) / linkLengthLeft;

	double linkLengthRight = computeDist(xLoc, yLoc, zLoc, xLocRight, yLocRight,
			zLocRight);
	double forceValueRight = sceProfilePara[5]
			* (linkLengthRight - sceProfilePara[6]);
	xVel = xVel + forceValueRight * (xLocRight - xLoc) / linkLengthRight;
	yVel = yVel + forceValueRight * (yLocRight - yLoc) / linkLengthRight;
	zVel = zVel + forceValueRight * (zLocRight - zLoc) / linkLengthRight;

}

__device__
void handleSceForceNodesBasic(uint& nodeRank1, uint& nodeRank2, double& xPos,
		double& yPos, double& zPos, double& xPos2, double& yPos2, double& zPos2,
		double& xRes, double& yRes, double& zRes, double* _nodeLocXAddress,
		double* _nodeLocYAddress, double* _nodeLocZAddress) {
	if (isSameCell(nodeRank1, nodeRank2)) {
		calculateAndAddIntraForce(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2], xRes,
				yRes, zRes);
	} else {
		calculateAndAddInterForce(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2], xRes,
				yRes, zRes);
	}
}

__device__
void handleSceForceNodesDisc(uint& nodeRank1, uint& nodeRank2, double& xPos,
		double& yPos, double& zPos, double& xPos2, double& yPos2, double& zPos2,
		double& xRes, double& yRes, double& zRes, double& interForceX,
		double& interForceY, double& interForceZ, double* _nodeLocXAddress,
		double* _nodeLocYAddress, double* _nodeLocZAddress,
		double* _nodeGrowProAddr) {
	if (isSameCell(nodeRank1, nodeRank2)) {
		calAndAddIntraForceDiv(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2],
				_nodeGrowProAddr[nodeRank2], xRes, yRes, zRes);
	} else {
		calAndAddInterForceDisc(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2], xRes,
				yRes, zRes, interForceX, interForceY, interForceZ);
	}
}

__device__
void handleSceForceNodesDisc_M(uint& nodeRank1, uint& nodeRank2, double& xPos,
		double& yPos, double& xPos2, double& yPos2, double& xRes, double& yRes,
		double* _nodeLocXAddress, double* _nodeLocYAddress,
		double* _nodeGrowProAddr) {
	if (isSameCell_m(nodeRank1, nodeRank2)) {
		if (bothInternal(nodeRank1, nodeRank2)) {
			// both nodes are internal type.
			calAndAddIntraDiv_M(xPos, yPos, _nodeLocXAddress[nodeRank2],
					_nodeLocYAddress[nodeRank2], _nodeGrowProAddr[nodeRank2],
					xRes, yRes);
		} else if (bothEpi(nodeRank1, nodeRank2)) {
			// both nodes epithilium type. no sce force applied.
			// nothing to do here.
		} else {
			// one node is epithilium type the other is internal type.
			calAndAddIntraB_M(xPos, yPos, _nodeLocXAddress[nodeRank2],
					_nodeLocYAddress[nodeRank2], xRes, yRes);
		}
	} else {
		if (bothEpi(nodeRank1, nodeRank2)) {
			calAndAddInter_M(xPos, yPos, _nodeLocXAddress[nodeRank2],
					_nodeLocYAddress[nodeRank2], xRes, yRes);
		}
	}
}

__device__
void handleForceBetweenNodes(uint &nodeRank1, SceNodeType &type1,
		uint &nodeRank2, SceNodeType &type2, double &xPos, double &yPos,
		double &zPos, double &xPos2, double &yPos2, double &zPos2, double &xRes,
		double &yRes, double &zRes, double &maxForce, double* _nodeLocXAddress,
		double* _nodeLocYAddress, double* _nodeLocZAddress) {
// this means that both nodes are come from cells, not other types
	if (bothCellNodes(type1, type2)) {
		// this means that nodes come from different type of cell, apply differential adhesion
		if (type1 != type2) {
			// differential adhesion applies here.
			calculateAndAddDiffInterCellForce(xPos, yPos, zPos,
					_nodeLocXAddress[nodeRank2], _nodeLocYAddress[nodeRank2],
					_nodeLocZAddress[nodeRank2], xRes, yRes, zRes);
		} else {
			if (isSameCell(nodeRank1, nodeRank2)) {
				calculateAndAddIntraForce(xPos, yPos, zPos,
						_nodeLocXAddress[nodeRank2],
						_nodeLocYAddress[nodeRank2],
						_nodeLocZAddress[nodeRank2], xRes, yRes, zRes);
			} else {
				double xPre = xRes;
				double yPre = yRes;
				double zPre = zRes;
				calculateAndAddInterForce(xPos, yPos, zPos,
						_nodeLocXAddress[nodeRank2],
						_nodeLocYAddress[nodeRank2],
						_nodeLocZAddress[nodeRank2], xRes, yRes, zRes);
				double xDiff = xRes - xPre;
				double yDiff = yRes - yPre;
				double zDiff = zRes - zPre;
				double force = sqrt(
						xDiff * xDiff + yDiff * yDiff + zDiff * zDiff);
				if (force > maxForce) {
					maxForce = force;
				}
			}
		}
	}

// this means that both nodes come from ECM and from same ECM
	else if (type1 == ECM && type2 == ECM && isSameECM(nodeRank1, nodeRank2)) {
		if (isNeighborECMNodes(nodeRank1, nodeRank2)) {
			calculateAndAddECMForce(xPos, yPos, zPos,
					_nodeLocXAddress[nodeRank2], _nodeLocYAddress[nodeRank2],
					_nodeLocZAddress[nodeRank2], xRes, yRes, zRes);
		}
		// if both nodes belong to same ECM but are not neighbors they shouldn't interact.
	} else if ((type1 == Profile && type2 == Cart)
			|| (type1 == Cart && type2 == Profile)) {
		calculateAndAddCartForce(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2], xRes,
				yRes, zRes);
	} else if (type1 == Cart && type2 == Cart) {
	} else if (type1 == Profile && type2 == Profile) {
	} else {
		// for now, we assume that interaction between other nodes are the same as inter-cell force.
		calculateAndAddInterForce(xPos, yPos, zPos, _nodeLocXAddress[nodeRank2],
				_nodeLocYAddress[nodeRank2], _nodeLocZAddress[nodeRank2], xRes,
				yRes, zRes);
	}

}

void SceNodes::extendBuckets2D() {
	static const uint extensionFactor2D = 9;
	uint valuesCount = auxVecs.bucketValues.size();
	auxVecs.bucketKeysExpanded.resize(valuesCount * extensionFactor2D);
	auxVecs.bucketValuesIncludingNeighbor.resize(
			valuesCount * extensionFactor2D);

	/**
	 * beginning of constant iterator
	 */
	thrust::constant_iterator<uint> first(extensionFactor2D);
	/**
	 * end of constant iterator.
	 * the plus sign only indicate movement of position, not value.
	 * e.g. movement is 5 and first iterator is initialized as 9
	 * result array is [9,9,9,9,9];
	 */
	thrust::constant_iterator<uint> last = first + valuesCount;

	expand(first, last,
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeys.begin(),
							auxVecs.bucketValues.begin())),
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeysExpanded.begin(),
							auxVecs.bucketValuesIncludingNeighbor.begin())));

	thrust::counting_iterator<uint> countingBegin(0);
	thrust::counting_iterator<uint> countingEnd = countingBegin
			+ valuesCount * extensionFactor2D;

	thrust::transform(
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeysExpanded.begin(),
							countingBegin)),
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeysExpanded.end(), countingEnd)),
			make_zip_iterator(
					make_tuple(auxVecs.bucketKeysExpanded.begin(),
							countingBegin)),
			NeighborFunctor2D(domainPara.numOfBucketsInXDim,
					domainPara.numOfBucketsInYDim));

	int numberOfOutOfRange = thrust::count(auxVecs.bucketKeysExpanded.begin(),
			auxVecs.bucketKeysExpanded.end(), UINT_MAX);

	int sizeBeforeShrink = auxVecs.bucketKeysExpanded.size();
	int numberInsideRange = sizeBeforeShrink - numberOfOutOfRange;
	thrust::sort_by_key(auxVecs.bucketKeysExpanded.begin(),
			auxVecs.bucketKeysExpanded.end(),
			auxVecs.bucketValuesIncludingNeighbor.begin());
	auxVecs.bucketKeysExpanded.erase(
			auxVecs.bucketKeysExpanded.begin() + numberInsideRange,
			auxVecs.bucketKeysExpanded.end());
	auxVecs.bucketValuesIncludingNeighbor.erase(
			auxVecs.bucketValuesIncludingNeighbor.begin() + numberInsideRange,
			auxVecs.bucketValuesIncludingNeighbor.end());
}

void SceNodes::applySceForcesBasic() {
	uint* valueAddress = thrust::raw_pointer_cast(
			&auxVecs.bucketValuesIncludingNeighbor[0]);
	double* nodeLocXAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocX[0]);
	double* nodeLocYAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocY[0]);
	double* nodeLocZAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocZ[0]);

	thrust::transform(
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.begin()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.begin()),
							auxVecs.bucketValues.begin(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.begin()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.end()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.end()),
							auxVecs.bucketValues.end(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.end()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(infoVecs.nodeVelX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelZ.begin(),
									auxVecs.bucketValues.begin()))),
			AddSceForceBasic(valueAddress, nodeLocXAddress, nodeLocYAddress,
					nodeLocZAddress));
}

void SceNodes::applySceForcesDisc() {
	uint* valueAddress = thrust::raw_pointer_cast(
			&auxVecs.bucketValuesIncludingNeighbor[0]);
	double* nodeLocXAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocX[0]);
	double* nodeLocYAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocY[0]);
	double* nodeLocZAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocZ[0]);
	double* nodeGrowProAddr = thrust::raw_pointer_cast(
			&infoVecs.nodeGrowPro[0]);

	thrust::transform(
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.begin()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.begin()),
							auxVecs.bucketValues.begin(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.begin()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.end()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.end()),
							auxVecs.bucketValues.end(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.end()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(infoVecs.nodeVelX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelZ.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(
									infoVecs.nodeInterForceX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(
									infoVecs.nodeInterForceY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(
									infoVecs.nodeInterForceZ.begin(),
									auxVecs.bucketValues.begin()))),
			AddSceForceDisc(valueAddress, nodeLocXAddress, nodeLocYAddress,
					nodeLocZAddress, nodeGrowProAddr));
}

void SceNodes::applySceForcesDisc_M() {
	uint* valueAddress = thrust::raw_pointer_cast(
			&auxVecs.bucketValuesIncludingNeighbor[0]);
	double* nodeLocXAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocX[0]);
	double* nodeLocYAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocY[0]);
	int* nodeAdhIdxAddress = thrust::raw_pointer_cast(
			&infoVecs.nodeAdhereIndex[0]);
	double* nodeGrowProAddr = thrust::raw_pointer_cast(
			&infoVecs.nodeGrowPro[0]);

	thrust::transform(
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.begin()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.begin()),
							auxVecs.bucketValues.begin(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.begin()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.end()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.end()),
							auxVecs.bucketValues.end(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.end()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(infoVecs.nodeVelX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelY.begin(),
									auxVecs.bucketValues.begin()))),
			AddForceDisc_M(valueAddress, nodeLocXAddress, nodeLocYAddress,
					nodeAdhIdxAddress, nodeGrowProAddr));
}

void SceNodes::applySceForces() {

// There are two reasons why I use thrust cast every time.
// (1) Technically, make a device pointer a global variable seems to be difficult.
// (2) Vectors might change the memory address dynamically.
	uint* valueAddress = thrust::raw_pointer_cast(
			&auxVecs.bucketValuesIncludingNeighbor[0]);
	double* nodeLocXAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocX[0]);
	double* nodeLocYAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocY[0]);
	double* nodeLocZAddress = thrust::raw_pointer_cast(&infoVecs.nodeLocZ[0]);

	SceNodeType* nodeTypeAddress = thrust::raw_pointer_cast(
			&infoVecs.nodeCellType[0]);

	thrust::transform(
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.begin()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.begin()),
							auxVecs.bucketValues.begin(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.begin()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(auxVecs.keyBegin.begin(),
									auxVecs.bucketKeys.end()),
							make_permutation_iterator(auxVecs.keyEnd.begin(),
									auxVecs.bucketKeys.end()),
							auxVecs.bucketValues.end(),
							make_permutation_iterator(infoVecs.nodeLocX.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocY.begin(),
									auxVecs.bucketValues.end()),
							make_permutation_iterator(infoVecs.nodeLocZ.begin(),
									auxVecs.bucketValues.end()))),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(infoVecs.nodeVelX.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelY.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(infoVecs.nodeVelZ.begin(),
									auxVecs.bucketValues.begin()),
							make_permutation_iterator(
									infoVecs.nodeMaxForce.begin(),
									auxVecs.bucketValues.begin()))),
			AddSceForce(valueAddress, nodeLocXAddress, nodeLocYAddress,
					nodeLocZAddress, nodeTypeAddress));
}

void SceNodes::calculateAndApplySceForces() {
	prepareSceForceComputation();
	applySceForces();
	applyProfileForces();
}

const SceDomainPara& SceNodes::getDomainPara() const {
	return domainPara;
}

void SceNodes::setDomainPara(const SceDomainPara& domainPara) {
	this->domainPara = domainPara;
}

const NodeAllocPara& SceNodes::getAllocPara() const {
	return allocPara;
}

void SceNodes::setAllocPara(const NodeAllocPara& allocPara) {
	this->allocPara = allocPara;
}

const NodeAuxVecs& SceNodes::getAuxVecs() const {
	return auxVecs;
}

void SceNodes::setAuxVecs(const NodeAuxVecs& auxVecs) {
	this->auxVecs = auxVecs;
}

NodeInfoVecs& SceNodes::getInfoVecs() {
	return infoVecs;
}

std::vector<std::vector<int> > SceNodes::obtainLabelMatrix(
		PixelizePara& pixelPara) {
	std::vector<std::vector<int> > result;
	std::vector<NodeWithLabel> nodeLabels;
	ResAnalysisHelper resHelper;
	resHelper.setPixelPara(pixelPara);

	thrust::host_vector<double> hostTmpVectorLocX = infoVecs.nodeLocX;
	thrust::host_vector<double> hostTmpVectorLocY = infoVecs.nodeLocY;
	thrust::host_vector<double> hostTmpVectorLocZ = infoVecs.nodeLocZ;
	thrust::host_vector<SceNodeType> hostTmpVectorNodeType =
			infoVecs.nodeCellType;
	thrust::host_vector<uint> hostTmpVectorNodeRank = infoVecs.nodeCellRank;
	thrust::host_vector<uint> hostTmpVectorIsActive = infoVecs.nodeIsActive;

	uint startIndex = allocPara.startPosCells;
	uint endIndex = startIndex
			+ allocPara.currentActiveCellCount * allocPara.maxNodeOfOneCell;
	for (uint i = startIndex; i < endIndex; i++) {
		if (hostTmpVectorIsActive[i] == true) {
			NodeWithLabel nodeLabel;
			nodeLabel.cellRank = hostTmpVectorNodeRank[i];
			nodeLabel.position = CVector(hostTmpVectorLocX[i],
					hostTmpVectorLocY[i], hostTmpVectorLocZ[i]);
			nodeLabels.push_back(nodeLabel);
		}
	}

	result = resHelper.outputLabelMatrix(nodeLabels);
	return result;
}

void SceNodes::processCartGrowthDir(CVector dir) {
	double growthDir[3];
	dir = dir.getUnitVector();
	growthDir[0] = dir.GetX();
	growthDir[1] = dir.GetY();
	growthDir[2] = dir.GetZ();
	hipMemcpyToSymbol(HIP_SYMBOL(cartGrowDirVec), growthDir, 3 * sizeof(double));
}

void SceNodes::initControlPara(bool isStab) {
	int simuTypeConfigValue =
			globalConfigVars.getConfigValue("SimulationType").toInt();
	controlPara.simuType = parseTypeFromConfig(simuTypeConfigValue);
	controlPara.controlSwitchs.outputBmpImg = globalConfigVars.getSwitchState(
			"Switch_OutputBMP");
	controlPara.controlSwitchs.outputLabelMatrix =
			globalConfigVars.getSwitchState("Switch_OutputLabelMatrix");
	controlPara.controlSwitchs.outputStat = globalConfigVars.getSwitchState(
			"Switch_OutputStat");
	controlPara.controlSwitchs.outputVtkFile = globalConfigVars.getSwitchState(
			"Switch_OutputVtk");
	if (isStab) {
		controlPara.controlSwitchs.stab = ON;
	} else {
		controlPara.controlSwitchs.stab = OFF;
	}

}

void SceNodes::sceForcesPerfTesting() {
	prepareSceForceComputation();
	applySceForcesBasic();
}

void SceNodes::sceForcesDisc() {
	prepareSceForceComputation();
	applySceForcesDisc();
}

void SceNodes::sceForcesDisc_M() {
	prepareSceForceComputation();
	//applySceForcesDisc_M();
}

double SceNodes::getMaxEffectiveRange() {
	double interLinkEffectiveRange = globalConfigVars.getConfigValue(
			"InterCellLinkEffectRange").toDouble();
	double maxEffectiveRange = interLinkEffectiveRange;

	double intraLinkEffectiveRange = globalConfigVars.getConfigValue(
			"IntraCellLinkEffectRange").toDouble();
	if (intraLinkEffectiveRange > maxEffectiveRange) {
		maxEffectiveRange = intraLinkEffectiveRange;
	}

	double cartEffectiveRange = 0;
// cartilage effective range does not apply for other types of simulation.
	try {
		cartEffectiveRange = globalConfigVars.getConfigValue(
				"CartForceEffectiveRange").toDouble();
	} catch (SceException &exce) {

	}
	if (cartEffectiveRange > maxEffectiveRange) {
		maxEffectiveRange = cartEffectiveRange;
	}
	return maxEffectiveRange;
}

void SceNodes::setInfoVecs(const NodeInfoVecs& infoVecs) {
	this->infoVecs = infoVecs;
}

void SceNodes::allocSpaceForNodes(uint maxTotalNodeCount) {
	infoVecs.nodeLocX.resize(maxTotalNodeCount);
	infoVecs.nodeLocY.resize(maxTotalNodeCount);
	infoVecs.nodeLocZ.resize(maxTotalNodeCount);
	infoVecs.nodeVelX.resize(maxTotalNodeCount);
	infoVecs.nodeVelY.resize(maxTotalNodeCount);
	infoVecs.nodeVelZ.resize(maxTotalNodeCount);
	infoVecs.nodeMaxForce.resize(maxTotalNodeCount);
	infoVecs.nodeCellType.resize(maxTotalNodeCount);
	infoVecs.nodeCellRank.resize(maxTotalNodeCount);
	infoVecs.nodeIsActive.resize(maxTotalNodeCount);
	if (controlPara.simuType == Disc
			|| controlPara.simuType == SingleCellTest) {
		infoVecs.nodeGrowPro.resize(maxTotalNodeCount);
		infoVecs.nodeInterForceX.resize(maxTotalNodeCount);
		infoVecs.nodeInterForceY.resize(maxTotalNodeCount);
		infoVecs.nodeInterForceZ.resize(maxTotalNodeCount);
	}
}

void SceNodes::initNodeAllocPara(uint totalBdryNodeCount,
		uint maxProfileNodeCount, uint maxCartNodeCount, uint maxTotalECMCount,
		uint maxNodeInECM, uint maxTotalCellCount, uint maxNodeInCell) {
	allocPara.maxCellCount = maxTotalCellCount;
	allocPara.maxNodeOfOneCell = maxNodeInCell;
	allocPara.maxNodePerECM = maxNodeInECM;
	allocPara.maxECMCount = maxTotalECMCount;
	allocPara.maxProfileNodeCount = maxProfileNodeCount;
	allocPara.maxCartNodeCount = maxCartNodeCount;

	allocPara.currentActiveProfileNodeCount = 0;
	allocPara.currentActiveCartNodeCount = 0;
	allocPara.BdryNodeCount = totalBdryNodeCount;
	allocPara.currentActiveCellCount = 0;
	allocPara.maxTotalECMNodeCount = allocPara.maxECMCount
			* allocPara.maxNodePerECM;
	allocPara.currentActiveECM = 0;

	allocPara.maxTotalCellNodeCount = maxTotalCellCount
			* allocPara.maxNodeOfOneCell;

	allocPara.startPosProfile = totalBdryNodeCount;
	allocPara.startPosCart = allocPara.startPosProfile
			+ allocPara.maxProfileNodeCount;
	allocPara.startPosECM = allocPara.startPosCart + allocPara.maxCartNodeCount;
	allocPara.startPosCells = allocPara.startPosECM
			+ allocPara.maxTotalECMNodeCount;
}

void SceNodes::initNodeAllocPara_M(uint totalBdryNodeCount,
		uint maxTotalCellCount, uint maxEpiNodePerCell,
		uint maxInternalNodePerCell) {
	allocPara_M.bdryNodeCount = totalBdryNodeCount;
	allocPara_M.currentActiveCellCount = 0;
	allocPara_M.maxCellCount = maxTotalCellCount;
	allocPara_M.maxAllNodePerCell = maxEpiNodePerCell + maxInternalNodePerCell;
	allocPara_M.maxMembrNodePerCell = maxEpiNodePerCell;
	allocPara_M.maxIntnlNodePerCell = maxInternalNodePerCell;
	allocPara_M.maxTotalNodeCount = allocPara_M.maxAllNodePerCell
			* allocPara_M.maxCellCount;
}

void SceNodes::removeNodes(int cellRank, vector<uint> &removeSeq) {
	uint cellBeginIndex = allocPara.startPosCells
			+ cellRank * allocPara.maxNodeOfOneCell;
	uint cellEndIndex = cellBeginIndex + allocPara.maxNodeOfOneCell;
	thrust::host_vector<double> cellXCoords(allocPara.maxNodeOfOneCell);
	thrust::host_vector<double> cellYCoords(allocPara.maxNodeOfOneCell);
	thrust::copy(infoVecs.nodeLocX.begin() + cellBeginIndex,
			infoVecs.nodeLocX.begin() + cellEndIndex, cellXCoords.begin());
	thrust::copy(infoVecs.nodeLocY.begin() + cellBeginIndex,
			infoVecs.nodeLocY.begin() + cellEndIndex, cellYCoords.begin());
	vector<bool> isRemove(allocPara.maxNodeOfOneCell, false);

	/*
	 std::cout << "before, X: [";
	 for (uint i = 0; i < allocPara.maxNodeOfOneCell; i++) {
	 std::cout << cellXCoords[i] << " ";
	 }
	 std::cout << "]" << endl;
	 std::cout << "before, Y: [";
	 for (uint i = 0; i < allocPara.maxNodeOfOneCell; i++) {
	 std::cout << cellYCoords[i] << " ";
	 }
	 std::cout << "]" << endl;
	 */

	for (uint i = 0; i < removeSeq.size(); i++) {
		isRemove[removeSeq[i]] = true;
	}
	thrust::host_vector<double> cellXRemoved(allocPara.maxNodeOfOneCell);
	thrust::host_vector<double> cellYRemoved(allocPara.maxNodeOfOneCell);
	uint curIndex = 0;
	for (uint i = 0; i < allocPara.maxNodeOfOneCell; i++) {
		if (isRemove[i] == false) {
			cellXRemoved[curIndex] = cellXCoords[i];
			cellYRemoved[curIndex] = cellYCoords[i];
			curIndex++;
		}
	}

	/*
	 std::cout << "after, X: [";
	 for (uint i = 0; i < allocPara.maxNodeOfOneCell; i++) {
	 std::cout << cellXRemoved[i] << " ";
	 }
	 std::cout << "]" << endl;
	 std::cout << "after, Y: [";
	 for (uint i = 0; i < allocPara.maxNodeOfOneCell; i++) {
	 std::cout << cellYRemoved[i] << " ";
	 }
	 std::cout << "]" << endl;
	 */
	thrust::copy(cellXRemoved.begin(), cellXRemoved.end(),
			infoVecs.nodeLocX.begin() + cellBeginIndex);
	thrust::copy(cellYRemoved.begin(), cellYRemoved.end(),
			infoVecs.nodeLocY.begin() + cellBeginIndex);
}
