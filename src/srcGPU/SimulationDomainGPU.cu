#include "hip/hip_runtime.h"
/**
 * @file SimulationDomainGPU.cu
 * @brief this file contains domain level logic.
 * @author Wenzhao Sun wsun2@nd.edu
 * @bug no know bugs
 */

#include "SimulationDomainGPU.h"

using namespace std;

/**
 * Constructor.
 * reads values from config file.
 */
SimulationDomainGPU::SimulationDomainGPU() {
	readAllParameters();
	if (memPara.simuType == Beak) {
		initializeGrowthMap();
	}
}

void SimulationDomainGPU::initializeNodes(CartPara &cartPara,
		std::vector<SceNodeType>& cellTypes,
		std::vector<uint>& numOfInitActiveNodesOfCells,
		std::vector<CVector>& initBdryNodeVec,
		std::vector<CVector>& initProfileNodeVec,
		std::vector<CVector>& initCartNodeVec,
		std::vector<CVector>& initECMNodeVec,
		std::vector<CVector>& initFNMNodeVec,
		std::vector<CVector>& initMXNodeVec) {

	/*
	 * number of boundary nodes is always fixed.
	 */
	uint bdryNodeCount = initBdryNodeVec.size();
	/*
	 * total potential number of profile nodes could be more than initial number provided,
	 * because the total length of profile might increase.
	 * FinalToInitProfileNodeCountRatio is defined in Config file.
	 */
	uint maxProfileNodeCount = initProfileNodeVec.size()
			* memPara.FinalToInitProfileNodeCountRatio;

	/**
	 * different from profile nodes, the cartilage nodes should already
	 * have buffer before executing this method.
	 */
	uint maxCartNodeCount = initCartNodeVec.size();

	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(bdryNodeCount, maxProfileNodeCount, maxCartNodeCount,
			memPara.maxECMInDomain, memPara.maxNodePerECM,
			memPara.maxCellInDomain, memPara.maxNodePerCell, memPara.isStab);

	//cartilage.distributeIsActive();
	//cartilage.initializeNodes(initCartNodeVec);

	/*
	 * first step: error checking.
	 * we need to first check if inputs are valid
	 */
	// get max node per cell. should be defined previously.
	uint maxNodePerCell = nodes.getAllocPara().maxNodeOfOneCell;
	// get max node per ECM. Should be defined previously.
	uint maxNodePerECM = nodes.getAllocPara().maxNodePerECM;
	// check if we successfully loaded maxNodePerCell
	assert(maxNodePerCell != 0);

	// obtain sizes of the input arrays
	//uint bdryNodeCount = initBdryNodeVec.size();
	uint ProfileNodeCount = initProfileNodeVec.size();
	uint CartNodeCount = initCartNodeVec.size();
	uint ECMNodeCount = initECMNodeVec.size();
	uint FNMNodeCount = initFNMNodeVec.size();
	uint MXNodeCount = initMXNodeVec.size();

	// array size of cell type array
	uint cellTypeSize = cellTypes.size();
	// array size of initial active node count of cells array.
	uint initNodeCountSize = numOfInitActiveNodesOfCells.size();
	// two sizes must match.
	assert(cellTypeSize == initNodeCountSize);

	// size of inputs must be divided exactly by max node per cell.
	// uint bdryRemainder = bdryNodeCountX % maxNodePerCell;
	uint ecmRemainder = 0;
	uint ecmQuotient = 0;
	if (memPara.simuType == Beak) {
		ecmQuotient = ECMNodeCount / maxNodePerECM;
		ecmRemainder = ECMNodeCount % maxNodePerECM;
	}
	uint fnmRemainder = FNMNodeCount % maxNodePerCell;
	uint mxRemainder = MXNodeCount % maxNodePerCell;

	// uint bdryQuotient = bdryNodeCountX / maxNodePerCell;
	uint fnmQuotient = FNMNodeCount / maxNodePerCell;
	uint mxQuotient = MXNodeCount / maxNodePerCell;

	// remainder must be zero.
	if ((fnmRemainder != 0) || (mxRemainder != 0) || (ecmRemainder != 0)) {
		throw SceException("Initialization vector size incorrect!",
				InputInitException);
	}
	// size of cellType array and sum of all cell types must match.
	assert(fnmQuotient + mxQuotient == cellTypeSize);

	for (uint i = 0; i < cellTypeSize; i++) {
		if (i < fnmQuotient) {
			assert(cellTypes[i] == FNM);
		} else {
			assert(cellTypes[i] == MX);
		}
	}
	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */
	NodeAllocPara para = nodes.getAllocPara();
	para.currentActiveCellCount = fnmQuotient + mxQuotient;
	para.currentActiveECM = ecmQuotient;
	para.currentActiveProfileNodeCount = ProfileNodeCount;
	nodes.setAllocPara(para);

	assert(nodes.getAllocPara().startPosProfile == bdryNodeCount);

	nodes.initValues_v2(initBdryNodeVec, initProfileNodeVec, initCartNodeVec,
			initECMNodeVec, initFNMNodeVec, initMXNodeVec);

	/**
	 * setting the cartilage related parameters in the simulation domain.
	 */
	if (memPara.simuType == Beak && !memPara.isStab) {
		cartilage.setCartPara(cartPara);
		cartilage.initializeMem(&nodes);
	}

	cells = SceCells(&nodes, numOfInitActiveNodesOfCells, cellTypes);
}

void SimulationDomainGPU::initialize_v2(SimulationInitData_V2& initData) {
	std::cout << "begin initialization process" << std::endl;
	memPara.isStab = initData.isStab;
	initializeNodes(initData.cartPara, initData.cellTypes,
			initData.numOfInitActiveNodesOfCells, initData.initBdryNodeVec,
			initData.initProfileNodeVec, initData.initCartNodeVec,
			initData.initECMNodeVec, initData.initFNMNodeVec,
			initData.initMXNodeVec);
	std::cout << "finished init simulation domain nodes" << std::endl;
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
	std::cout << "finished init nodes dimension" << std::endl;
	// The domain task is not stabilization unless specified in the next steps.
	stabPara.isProcessStab = false;
}

/**
 * Highest level logic of domain.
 *
 */
void SimulationDomainGPU::runAllLogic(double dt) {
	if (memPara.simuType == Beak && !stabPara.isProcessStab) {
		nodes.processCartGrowthDir(cartilage.getCartPara().growthDir);
		//std::cout << "growth direction is ";
		//cartilage.getCartPara().growthDir.Print();
	}

	// This function only calculates velocity.
	nodes.calculateAndApplySceForces();
	// Only beak simulation need to take care of cartilage.
	if (memPara.simuType == Beak && !stabPara.isProcessStab) {
		// cartilage logics must come before cell logics, because node velocities will be modified
		// in cell logic and consequently we won't be able to compute cartilage data.
		// also responsible for handling interaction between epithelium layer and carilage.
		cartilage.runAllLogics(dt);
	}
	// This function applies velocity so nodes actually move inside this function.
	if (memPara.simuType == Beak) {
		cells.runAllCellLevelLogicsBeak(dt, growthMap, growthMap2);
	} else if (memPara.simuType == Disc) {
		cells.runAllCellLevelLogicsDisc(dt);
	}
}

void SimulationDomainGPU::readMemPara() {
	int simuTypeConfigValue =
			globalConfigVars.getConfigValue("SimulationType").toInt();
	if (simuTypeConfigValue == 0) {
		memPara.simuType = Beak;
	} else if (simuTypeConfigValue == 1) {
		memPara.simuType = Disc;
	} else {
		throw SceException("Simulation Type in config file is not recognized!",
				ConfigValueException);
	}

	memPara.maxCellInDomain =
			globalConfigVars.getConfigValue("MaxCellInDomain").toInt();
	memPara.maxNodePerCell =
			globalConfigVars.getConfigValue("MaxNodePerCell").toInt();
	if (memPara.simuType == Beak) {
		memPara.maxECMInDomain = globalConfigVars.getConfigValue(
				"MaxECMInDomain").toInt();
		memPara.maxNodePerECM =
				globalConfigVars.getConfigValue("MaxNodePerECM").toInt();
		memPara.FinalToInitProfileNodeCountRatio =
				globalConfigVars.getConfigValue(
						"FinalToInitProfileNodeCountRatio").toDouble();
		//memPara.FinalToInitCartNodeCountRatio = globalConfigVars.getConfigValue(
		//		"FinalToInitCartNodeCountRatio").toDouble();
	} else {
		memPara.maxECMInDomain = 0;
		memPara.maxNodePerECM = 0;
		memPara.FinalToInitProfileNodeCountRatio = 0;
	}
}

void SimulationDomainGPU::readDomainPara() {
	domainPara.minX = globalConfigVars.getConfigValue("DOMAIN_XMIN").toDouble();
	domainPara.maxX = globalConfigVars.getConfigValue("DOMAIN_XMAX").toDouble();
	domainPara.minY = globalConfigVars.getConfigValue("DOMAIN_YMIN").toDouble();
	domainPara.maxY = globalConfigVars.getConfigValue("DOMAIN_YMAX").toDouble();
	domainPara.minZ = globalConfigVars.getConfigValue("DOMAIN_ZMIN").toDouble();
	domainPara.maxZ = globalConfigVars.getConfigValue("DOMAIN_ZMAX").toDouble();
	domainPara.gridSpacing = nodes.getMaxEffectiveRange();
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInYDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
}

void SimulationDomainGPU::readChemPara() {
	chemPara.growthGridXDim =
			globalConfigVars.getConfigValue("GrowthGridXDim").toInt();
	chemPara.growthGridYDim =
			globalConfigVars.getConfigValue("GrowthGridYDim").toInt();
	chemPara.growthGridSpacing = globalConfigVars.getConfigValue(
			"GrowthGridSpacing").toDouble();
	chemPara.growthGridLowerLeftPtX = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtX").toDouble();
	chemPara.growthGridLowerLeftPtY = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtY").toDouble();

	chemPara.growthMorCenterXCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoord").toDouble();
	chemPara.growthMorCenterYCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoord").toDouble();
	chemPara.growthMorHighConcen = globalConfigVars.getConfigValue(
			"GrowthMorHighConcen").toDouble();
	chemPara.growthMorLowConcen = globalConfigVars.getConfigValue(
			"GrowthMorLowConcen").toDouble();
	chemPara.growthMorDiffSlope = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlope").toDouble();

	chemPara.growthMorCenterXCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoordMX").toDouble();
	chemPara.growthMorCenterYCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoordMX").toDouble();
	chemPara.growthMorHighConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorHighConcenMX").toDouble();
	chemPara.growthMorLowConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorLowConcenMX").toDouble();
	chemPara.growthMorDiffSlopeMX = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlopeMX").toDouble();
}

void SimulationDomainGPU::readAllParameters() {
	readMemPara();
	readDomainPara();
	if (memPara.simuType == Beak) {
		readChemPara();
	}
}

void SimulationDomainGPU::initializeGrowthMap() {
	growthMap = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoord,
			chemPara.growthMorCenterYCoord, chemPara.growthMorHighConcen,
			chemPara.growthMorLowConcen, chemPara.growthMorDiffSlope);

	//cout << "after created growthMap1" << endl;
	growthMap2 = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap2.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoordMX,
			chemPara.growthMorCenterYCoordMX, chemPara.growthMorHighConcenMX,
			chemPara.growthMorLowConcenMX, chemPara.growthMorDiffSlopeMX);
	//cout << "after created growthMap2" << endl;
}

std::vector<CVector> SimulationDomainGPU::stablizeCellCenters(
		SimulationInitData_V2 &initData) {

	std::vector<CVector> result;

	stabPara.outputFrameCount = globalConfigVars.getConfigValue(
			"StabFrameCount").toInt();
	stabPara.totalIterCount = globalConfigVars.getConfigValue(
			"StabTotalIterCount").toInt();
	stabPara.bdrySpacingRatio = globalConfigVars.getConfigValue(
			"StabBdrySpacingRatio").toDouble();
	stabPara.dt = globalConfigVars.getConfigValue("StabDt").toDouble();
	stabPara.outputAniName =
			globalConfigVars.getConfigValue("StabAniName").toString();

	initialize_v2(initData);
	stabPara.isProcessStab = true;
	int aniAuxPara;
	if (stabPara.outputFrameCount == 0) {
		aniAuxPara = INT_MAX;
	} else {
		aniAuxPara = (double) (stabPara.totalIterCount)
				/ stabPara.outputFrameCount;
	}

	AnimationCriteria aniCri;
	aniCri.defaultEffectiveDistance = globalConfigVars.getConfigValue(
			"IntraLinkDisplayRange").toDouble();
	aniCri.isStressMap = false;

	uint index = 0;
	for (int i = 0; i < stabPara.totalIterCount; i++) {
		//std::cout << "in stablizing, before run all logics" << std::endl;
		if (i % aniAuxPara == 0) {
			outputVtkFilesWithColor(stabPara.outputAniName, index, aniCri);
			index++;
		}
		runAllLogic(stabPara.dt);
	}

	result = cells.getAllCellCenters();

	cout << "finished stablizeCellCenters" << endl;
	cout.flush();
	return result;
}

void SimulationDomainGPU::outputVtkFilesWithColor(std::string scriptNameBase,
		int rank, AnimationCriteria aniCri) {
	nodes.prepareSceForceComputation();
	VtkAnimationData aniData = nodes.obtainAnimationData(aniCri);
	aniData.outputVtkAni(scriptNameBase, rank);
}

void SimulationDomainGPU::printDomainInformation() {
	cout << "Begin output information about nodes:" << endl;
	cout << "size of isActive:" << nodes.getInfoVecs().nodeIsActive.size()
			<< endl;
	cout << "size of nodeLocX:" << nodes.getInfoVecs().nodeLocX.size() << endl;
	cout << "size of nodeLocY:" << nodes.getInfoVecs().nodeLocY.size() << endl;
	cout << "size of nodeLocZ:" << nodes.getInfoVecs().nodeLocZ.size() << endl;
	cout << "size of nodeVelX:" << nodes.getInfoVecs().nodeVelX.size() << endl;
	cout << "size of nodeVelY:" << nodes.getInfoVecs().nodeVelY.size() << endl;
	cout << "size of nodeVelZ:" << nodes.getInfoVecs().nodeVelZ.size() << endl;
	cout << "size of CellType:" << nodes.getInfoVecs().nodeCellType.size()
			<< endl;
	cout << "size of nodeCellRank:" << nodes.getInfoVecs().nodeCellRank.size()
			<< endl;

	cout << "start position of Profile is "
			<< nodes.getAllocPara().startPosProfile << endl;
	cout << "start position of ECM is " << nodes.getAllocPara().startPosECM
			<< endl;
	cout << "start position of Cells is " << nodes.getAllocPara().startPosCells
			<< endl;

	cout << "max node of one cell is " << nodes.getAllocPara().maxNodeOfOneCell
			<< endl;
	cout << "max number of cells is " << nodes.getAllocPara().maxCellCount
			<< endl;
	cout << "max total cell node count is "
			<< nodes.getAllocPara().maxTotalCellNodeCount << endl;
	cout << "current active cell count is "
			<< nodes.getAllocPara().currentActiveCellCount << endl;

	cout << "max node of one ECM is " << nodes.getAllocPara().maxNodePerECM
			<< endl;
	cout << "max number of ECm is " << nodes.getAllocPara().maxECMCount << endl;
	cout << "max total ECM node count is "
			<< nodes.getAllocPara().maxTotalECMNodeCount << endl;
	cout << "current active ECM count is "
			<< nodes.getAllocPara().currentActiveECM << endl;

	cout << "max profile node count is "
			<< nodes.getAllocPara().maxProfileNodeCount << endl;
	cout << "current active profile node count is "
			<< nodes.getAllocPara().currentActiveProfileNodeCount << endl;
}

vector<vector<int> > SimulationDomainGPU::outputLabelMatrix(
		std::string resultNameBase, int rank, PixelizePara& pixelPara) {
	std::stringstream ss;
	ss << std::setw(5) << std::setfill('0') << rank;
	std::string resultNameRank = ss.str();
	std::string matrixFileName = resultNameBase + resultNameRank + ".dat";
	vector<vector<int> > matrix = nodes.obtainLabelMatrix(pixelPara);
	printMatrixToFile(matrix, matrixFileName);
	return matrix;
}

void SimulationDomainGPU::analyzeLabelMatrix(vector<vector<int> > &labelMatrix,
		int step, std::string &imageFileNameBase, std::string &statFileName) {
	ResAnalysisHelper resHelper;

	std::stringstream ss;
	ss << std::setw(5) << std::setfill('0') << step;
	std::string imgNameRank = ss.str();
	std::string imgFileName = imageFileNameBase + imgNameRank + ".bmp";

	resHelper.outputImg_formatBMP(imgFileName, labelMatrix);
	resHelper.outputStat_PolygonCounting(statFileName, step, labelMatrix);
}
