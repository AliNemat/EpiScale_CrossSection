#include "hip/hip_runtime.h"
/**
 * @file SimulationDomainGPU.cu
 * @brief this file contains domain level logic.
 * @author Wenzhao Sun wsun2@nd.edu
 * @bug no know bugs
 */

#include "SimulationDomainGPU.h"

using namespace std;

/**
 * Constructor.
 * reads values from config file.
 */
SimulationDomainGPU::SimulationDomainGPU() {
	readAllParameters();
	initializeGrowthMap();
}

/**
 * Initialize five different cell nodes.
 * We have to initialize five types of cells:
 * first is Boundary (B), fixed nodes on the boundary;
 * second is Profile (P), Epithilum cells;
 * third is ECM (E), extra-cellular matrix;
 * fourth is FNM (F), front nasal mass;
 * fifth is MX (M) maxillary cells.
 *
 * like this:
 * B-B-B-B-B-B-B-B-B-P-P-P-P-E-E-E-E-F-F-F-F-F-F-F-F-M-M-M-M-M-M-M-M
 * B, P and E is fixed. F and M will grow.
 * Rules:
 * 1a, Number of boundary nodes is fixed.
 * 1b, Profile nodes may or may not increase. Still testing the model.
 *     however the space is always reserved for it, though some spaces may not be active.
 * 1c, Extra-cellular matrix will grow.
 *     however the space is always reserved for it, but some spaces are not active.
 * 1d, In F part, each input vector must be divided exactly by (max node per cell)
 * 1e, In M part, each input vector must be divided exactly by (max node per cell)
 * 2a, Sum of number of cells from init FNM and init MX input vectors must be size of cellTypes
 *     so that all cells will have its own type
 * 2b, Read number of node per cell, etc, from config file.
 * 3a, First part of this function is error checking.
 * 3b, Second part of this function is the actual initialization
 */
void SimulationDomainGPU::initialCellsOfFiveTypes(
		std::vector<SceNodeType> &cellTypes,
		std::vector<uint> &numOfInitActiveNodesOfCells,
		std::vector<double> &initBdryCellNodePosX,
		std::vector<double> &initBdryCellNodePosY,
		std::vector<double> &initProfileNodePosX,
		std::vector<double> &initProfileNodePosY,
		std::vector<double> &initECMNodePosX,
		std::vector<double> &initECMNodePosY,
		std::vector<double> &initFNMCellNodePosX,
		std::vector<double> &initFNMCellNodePosY,
		std::vector<double> &initMXCellNodePosX,
		std::vector<double> &initMXCellNodePosY) {

	/*
	 * zero step: redefine nodes.
	 */
	/*
	 * number of boundary nodes is always fixed.
	 */
	uint bdryNodeCount = initBdryCellNodePosX.size();
	// total potential number of profile nodes could be more than initial number provided,
	// because the total length of profile might increase.
	// FinalToInitProfileNodeCountRatio is defined in Config file.
	uint maxProfileNodeCount = initProfileNodePosX.size()
			* memPara.FinalToInitProfileNodeCountRatio;

	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(bdryNodeCount, maxProfileNodeCount, memPara.maxECMInDomain,
			memPara.maxNodePerECM, memPara.maxCellInDomain,
			memPara.maxNodePerCell);

	/*
	 * first step: error checking.
	 * we need to first check if inputs are valid
	 */
	cout << "begin init cells of five types" << endl;
	// get max node per cell. should be defined previously.
	uint maxNodePerCell = nodes.getAllocPara().maxNodeOfOneCell;
	// get max node per ECM. Should be defined previously.
	uint maxNodePerECM = nodes.getAllocPara().maxNodePerECM;
	// check if we successfully loaded maxNodePerCell
	assert(maxNodePerCell != 0);

	// obtain sizes of the input arrays
	uint bdryNodeCountX = initBdryCellNodePosX.size();
	uint bdryNodeCountY = initBdryCellNodePosY.size();
	uint ProfileNodeCountX = initProfileNodePosX.size();
	uint ProfileNodeCountY = initProfileNodePosY.size();
	uint ECMNodeCountX = initECMNodePosX.size();
	uint ECMNodeCountY = initECMNodePosY.size();
	uint FNMNodeCountX = initFNMCellNodePosX.size();
	uint FNMNodeCountY = initFNMCellNodePosY.size();
	uint MXNodeCountX = initMXCellNodePosX.size();
	uint MXNodeCountY = initMXCellNodePosY.size();

	cout << "size of all node vectors: Boundary = " << bdryNodeCountX
			<< ", init profile node count =" << "," << ProfileNodeCountX
			<< "max profile node count = " << maxProfileNodeCount
			<< ", initial ECM node count = " << ECMNodeCountX
			<< ", init FNM node count = " << FNMNodeCountX
			<< ", init MX node count = " << MXNodeCountX << endl;

	//int jj;
	//cin >> jj;

	// array size of cell type array
	uint cellTypeSize = cellTypes.size();
	// array size of initial active node count of cells array.
	uint initNodeCountSize = numOfInitActiveNodesOfCells.size();
	// two sizes must match.
	assert(cellTypeSize == initNodeCountSize);
	// size of X and Y must match.
	assert(bdryNodeCountX == bdryNodeCountY);
	assert(ECMNodeCountX == ECMNodeCountY);
	assert(ProfileNodeCountX == ProfileNodeCountY);
	assert(FNMNodeCountX == FNMNodeCountY);
	assert(MXNodeCountX == MXNodeCountY);

	cout << "passed init checks" << endl;

	// size of inputs must be divided exactly by max node per cell.
	// uint bdryRemainder = bdryNodeCountX % maxNodePerCell;
	uint ecmRemainder = ECMNodeCountX % maxNodePerECM;
	uint fnmRemainder = FNMNodeCountX % maxNodePerCell;
	uint mxRemainder = MXNodeCountX % maxNodePerCell;

	// uint bdryQuotient = bdryNodeCountX / maxNodePerCell;
	uint ecmQuotient = ECMNodeCountX / maxNodePerECM;
	uint fnmQuotient = FNMNodeCountX / maxNodePerCell;
	uint mxQuotient = MXNodeCountX / maxNodePerCell;

	// for now we try to make boundary cells one complete part so ....
	//uint bdryRemainder = 0;
	uint bdryQuotient = 1;

	// for now we try to make profile nodes one complete part soremdiner = 0 and quotient = 1
	//uint profileRemainder = 0;
	uint profileQuotient = 1;

	// remainder must be zero.
	assert((fnmRemainder == 0) && (mxRemainder == 0) && (ecmRemainder == 0));
	// size of cellType array and sum of all cell types must match.
	assert(fnmQuotient + mxQuotient == cellTypeSize);

	cerr << "passed size assertion" << endl;

	// make sure the cell types follow format requirement.
	// must follow sequence : B - P - E - F - M
	int counter = 0;
	//SceNodeType cellTypesForEachLevel[5] = { Boundary, Profile, ECM, FNM, MX };
	int bounds[5];
	bounds[0] = bdryQuotient;
	bounds[1] = bounds[0] + profileQuotient;
	bounds[2] = bounds[1] + ecmQuotient;
	bounds[3] = bounds[2] + fnmQuotient;
	bounds[4] = bounds[3] + mxQuotient;
	int level = 0;
	while (counter < cellTypeSize) {
		// if count is already beyond the bound, we need to increase the current level.
		if (counter == bounds[level]) {
			level++;
		}
		// make sure that the input cell types array fits the calculated result.
		// depreciated -- requirement changed.
		// assert(cellTypes[counter] == cellTypesForEachLevel[level]);
		counter++;
	}
	cerr << "before set parameters" << endl;
	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */

	NodeAllocPara para = nodes.getAllocPara();
	para.currentActiveCellCount = fnmQuotient + mxQuotient;
	para.currentActiveECM = ecmQuotient;
	para.currentActiveProfileNodeCount = ProfileNodeCountX;
	nodes.setAllocPara(para);

	//NodeAllocPara nodePara = nodes.getAllocPara();

	assert(nodes.getAllocPara().startPosProfile == bdryNodeCountX);

	uint totalSize = nodes.getInfoVecs().nodeLocX.size();

	// set cell types
	//thrust::device_vector<SceNodeType> cellTypesToPass = cellTypes;

	nodes.initValues(initBdryCellNodePosX, initBdryCellNodePosY,
			initProfileNodePosX, initProfileNodePosY, initECMNodePosX,
			initECMNodePosY, initFNMCellNodePosX, initFNMCellNodePosY,
			initMXCellNodePosX, initMXCellNodePosY);

	/*
	 * Initialize SceCells_M ( M means modified) by nodes information.
	 */
	//cells = SceCells(&nodes);
	// copy initial active node count info to GPU
	//thrust::copy(numOfInitActiveNodesOfCells.begin(),
	//		numOfInitActiveNodesOfCells.end(),
	//		cells.activeNodeCountOfThisCell.begin());
	// set cell types
	//cells.setCellTypes(cellTypesToPass);
	//cells.distributeIsActiveInfo();
	cells = SceCells(&nodes, numOfInitActiveNodesOfCells, cellTypes);
}

void SimulationDomainGPU::initializeNodes(CartPara &cartPara,
		std::vector<SceNodeType>& cellTypes,
		std::vector<uint>& numOfInitActiveNodesOfCells,
		std::vector<CVector>& initBdryNodeVec,
		std::vector<CVector>& initProfileNodeVec,
		std::vector<CVector>& initCartNodeVec,
		std::vector<CVector>& initECMNodeVec,
		std::vector<CVector>& initFNMNodeVec,
		std::vector<CVector>& initMXNodeVec) {

	/*
	 * number of boundary nodes is always fixed.
	 */
	uint bdryNodeCount = initBdryNodeVec.size();
	/*
	 * total potential number of profile nodes could be more than initial number provided,
	 * because the total length of profile might increase.
	 * FinalToInitProfileNodeCountRatio is defined in Config file.
	 */
	uint maxProfileNodeCount = initProfileNodeVec.size()
			* memPara.FinalToInitProfileNodeCountRatio;

	/**
	 * different from profile nodes, the cartilage nodes should already
	 * have buffer before executing this method.
	 */
	uint maxCartNodeCount = initCartNodeVec.size();

	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(bdryNodeCount, maxProfileNodeCount, maxCartNodeCount,
			memPara.maxECMInDomain, memPara.maxNodePerECM,
			memPara.maxCellInDomain, memPara.maxNodePerCell);

	//cartilage.distributeIsActive();
	//cartilage.initializeNodes(initCartNodeVec);

	/*
	 * first step: error checking.
	 * we need to first check if inputs are valid
	 */
	// get max node per cell. should be defined previously.
	uint maxNodePerCell = nodes.getAllocPara().maxNodeOfOneCell;
	// get max node per ECM. Should be defined previously.
	uint maxNodePerECM = nodes.getAllocPara().maxNodePerECM;
	// check if we successfully loaded maxNodePerCell
	assert(maxNodePerCell != 0);

	// obtain sizes of the input arrays
	//uint bdryNodeCount = initBdryNodeVec.size();
	uint ProfileNodeCount = initProfileNodeVec.size();
	uint CartNodeCount = initCartNodeVec.size();
	uint ECMNodeCount = initECMNodeVec.size();
	uint FNMNodeCount = initFNMNodeVec.size();
	uint MXNodeCount = initMXNodeVec.size();

	// array size of cell type array
	uint cellTypeSize = cellTypes.size();
	// array size of initial active node count of cells array.
	uint initNodeCountSize = numOfInitActiveNodesOfCells.size();
	// two sizes must match.
	assert(cellTypeSize == initNodeCountSize);

	// size of inputs must be divided exactly by max node per cell.
	// uint bdryRemainder = bdryNodeCountX % maxNodePerCell;
	uint ecmRemainder = ECMNodeCount % maxNodePerECM;
	uint fnmRemainder = FNMNodeCount % maxNodePerCell;
	uint mxRemainder = MXNodeCount % maxNodePerCell;

	// uint bdryQuotient = bdryNodeCountX / maxNodePerCell;
	uint ecmQuotient = ECMNodeCount / maxNodePerECM;
	uint fnmQuotient = FNMNodeCount / maxNodePerCell;
	uint mxQuotient = MXNodeCount / maxNodePerCell;

	// remainder must be zero.
	assert((fnmRemainder == 0) && (mxRemainder == 0) && (ecmRemainder == 0));
	// size of cellType array and sum of all cell types must match.
	assert(fnmQuotient + mxQuotient == cellTypeSize);

	for (uint i = 0; i < cellTypeSize; i++) {
		if (i < fnmQuotient) {
			assert(cellTypes[i] == FNM);
		} else {
			assert(cellTypes[i] == MX);
		}
	}
	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */
	NodeAllocPara para = nodes.getAllocPara();
	para.currentActiveCellCount = fnmQuotient + mxQuotient;
	para.currentActiveECM = ecmQuotient;
	para.currentActiveProfileNodeCount = ProfileNodeCount;
	nodes.setAllocPara(para);

	assert(nodes.getAllocPara().startPosProfile == bdryNodeCount);

	nodes.initValues_v2(initBdryNodeVec, initProfileNodeVec, initCartNodeVec,
			initECMNodeVec, initFNMNodeVec, initMXNodeVec);

	/**
	 * setting the cartilage related parameters in the simulation domain.
	 */
	if (memPara.simuType == Beak) {
		cartilage.setCartPara(cartPara);
		cartilage.initializeMem(&nodes);
	}

	cells = SceCells(&nodes, numOfInitActiveNodesOfCells, cellTypes);
}

void SimulationDomainGPU::initialize(SimulationInitData &initData) {
	initialCellsOfFiveTypes(initData.cellTypes,
			initData.numOfInitActiveNodesOfCells, initData.initBdryCellNodePosX,
			initData.initBdryCellNodePosY, initData.initProfileNodePosX,
			initData.initProfileNodePosY, initData.initECMNodePosX,
			initData.initECMNodePosY, initData.initFNMCellNodePosX,
			initData.initFNMCellNodePosY, initData.initMXCellNodePosX,
			initData.initMXCellNodePosY);
	std::cout << "finished init cells of five types" << std::endl;
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
	std::cout << "finished init nodes dimension" << std::endl;
}

void SimulationDomainGPU::initialize_v2(SimulationInitData_V2& initData) {
	initializeNodes(initData.cartPara, initData.cellTypes,
			initData.numOfInitActiveNodesOfCells, initData.initBdryNodeVec,
			initData.initProfileNodeVec, initData.initCartNodeVec,
			initData.initECMNodeVec, initData.initFNMNodeVec,
			initData.initMXNodeVec);
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
}

/**
 * Highest level logic of domain.
 *
 */
void SimulationDomainGPU::runAllLogic(double dt) {
	if (memPara.simuType == Beak) {
		nodes.processCartGrowthDir(cartilage.getCartPara().growthDir);
		//std::cout << "growth direction is ";
		cartilage.getCartPara().growthDir.Print();
	}

	// This function only calculates velocity.
	nodes.calculateAndApplySceForces();
	// Only beak simulation need to take care of cartilage.
	if (memPara.simuType == Beak) {
		// cartilage logics must come before cell logics, because node velocities will be modified
		// in cell logic and consequently we won't be able to compute cartilage data.
		// also responsible for handling interaction between epithelium layer and carilage.
		cartilage.runAllLogics(dt);
	}
	// This function applies velocity so nodes actually move inside this function.
	cells.runAllCellLevelLogics(dt, growthMap, growthMap2);

}

void SimulationDomainGPU::readMemPara() {
	int simuTypeConfigValue =
			globalConfigVars.getConfigValue("SimulationType").toInt();
	if (simuTypeConfigValue == 0) {
		memPara.simuType = Beak;
	} else if (simuTypeConfigValue == 1) {
		memPara.simuType = Disc;
	} else {
		throw SceException("Simulation Type in config file is not recognized!",
				ConfigValueException);
	}

	memPara.maxCellInDomain =
			globalConfigVars.getConfigValue("MaxCellInDomain").toInt();
	memPara.maxNodePerCell =
			globalConfigVars.getConfigValue("MaxNodePerCell").toInt();
	memPara.maxECMInDomain =
			globalConfigVars.getConfigValue("MaxECMInDomain").toInt();
	memPara.maxNodePerECM =
			globalConfigVars.getConfigValue("MaxNodePerECM").toInt();
	memPara.FinalToInitProfileNodeCountRatio = globalConfigVars.getConfigValue(
			"FinalToInitProfileNodeCountRatio").toDouble();
	//memPara.FinalToInitCartNodeCountRatio = globalConfigVars.getConfigValue(
	//		"FinalToInitCartNodeCountRatio").toDouble();
}

void SimulationDomainGPU::readDomainPara() {
	domainPara.minX = globalConfigVars.getConfigValue("DOMAIN_XMIN").toDouble();
	domainPara.maxX = globalConfigVars.getConfigValue("DOMAIN_XMAX").toDouble();
	domainPara.minY = globalConfigVars.getConfigValue("DOMAIN_YMIN").toDouble();
	domainPara.maxY = globalConfigVars.getConfigValue("DOMAIN_YMAX").toDouble();
	domainPara.minZ = globalConfigVars.getConfigValue("DOMAIN_ZMIN").toDouble();
	domainPara.maxZ = globalConfigVars.getConfigValue("DOMAIN_ZMAX").toDouble();
	domainPara.gridSpacing = globalConfigVars.getConfigValue(
			"DOMAIN_GRID_SPACING").toDouble();
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInYDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
}

void SimulationDomainGPU::readChemPara() {
	chemPara.growthGridXDim =
			globalConfigVars.getConfigValue("GrowthGridXDim").toInt();
	chemPara.growthGridYDim =
			globalConfigVars.getConfigValue("GrowthGridYDim").toInt();
	chemPara.growthGridSpacing = globalConfigVars.getConfigValue(
			"GrowthGridSpacing").toDouble();
	chemPara.growthGridLowerLeftPtX = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtX").toDouble();
	chemPara.growthGridLowerLeftPtY = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtY").toDouble();

	chemPara.growthMorCenterXCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoord").toDouble();
	chemPara.growthMorCenterYCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoord").toDouble();
	chemPara.growthMorHighConcen = globalConfigVars.getConfigValue(
			"GrowthMorHighConcen").toDouble();
	chemPara.growthMorLowConcen = globalConfigVars.getConfigValue(
			"GrowthMorLowConcen").toDouble();
	chemPara.growthMorDiffSlope = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlope").toDouble();

	chemPara.growthMorCenterXCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoordMX").toDouble();
	chemPara.growthMorCenterYCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoordMX").toDouble();
	chemPara.growthMorHighConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorHighConcenMX").toDouble();
	chemPara.growthMorLowConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorLowConcenMX").toDouble();
	chemPara.growthMorDiffSlopeMX = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlopeMX").toDouble();
}

void SimulationDomainGPU::readAllParameters() {
	readMemPara();
	readDomainPara();
	readChemPara();
}

void SimulationDomainGPU::initializeGrowthMap() {
	growthMap = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoord,
			chemPara.growthMorCenterYCoord, chemPara.growthMorHighConcen,
			chemPara.growthMorLowConcen, chemPara.growthMorDiffSlope);

	//cout << "after created growthMap1" << endl;
	growthMap2 = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap2.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoordMX,
			chemPara.growthMorCenterYCoordMX, chemPara.growthMorHighConcenMX,
			chemPara.growthMorLowConcenMX, chemPara.growthMorDiffSlopeMX);
	//cout << "after created growthMap2" << endl;
}

std::vector<CVector> SimulationDomainGPU::stablizeCellCenters(
		SimulationInitData& initData) {

	std::vector<CVector> result;

	stabPara.outputFrameCount = globalConfigVars.getConfigValue(
			"StabFrameCount").toInt();
	stabPara.totalIterCount = globalConfigVars.getConfigValue(
			"StabTotalIterCount").toInt();
	stabPara.bdrySpacingRatio = globalConfigVars.getConfigValue(
			"StabBdrySpacingRatio").toDouble();
	stabPara.dt = globalConfigVars.getConfigValue("StabDt").toDouble();
	stabPara.outputAniName =
			globalConfigVars.getConfigValue("StabAniName").toString();

	initialize(initData);

	int aniAuxPara = (double) (stabPara.totalIterCount)
			/ stabPara.outputFrameCount;

	AnimationCriteria aniCri;
	aniCri.defaultEffectiveDistance = globalConfigVars.getConfigValue(
			"IntraLinkDisplayRange").toDouble();
	aniCri.isStressMap = false;

	for (int i = 0; i < stabPara.totalIterCount; i++) {
		//std::cout << "in stablizing, before run all logics" << std::endl;
		runAllLogic(stabPara.dt);
		if (i % aniAuxPara == 0) {
			outputVtkFilesWithColor(stabPara.outputAniName, i, aniCri);
		}
	}

	result = cells.getAllCellCenters();

	cout << "finished stablizeCellCenters" << endl;
	cout.flush();
	return result;
}

void SimulationDomainGPU::outputVtkFilesWithColor(std::string scriptNameBase,
		int rank, AnimationCriteria aniCri) {
	nodes.prepareSceForceComputation();
	VtkAnimationData aniData = nodes.obtainAnimationData(aniCri);
	aniData.outputVtkAni(scriptNameBase, rank);
}

void SimulationDomainGPU::checkIfAllDataFieldsValid() {
	cout << "Begin output information about nodes:" << endl;
	cout << "size of isActive:" << nodes.getInfoVecs().nodeIsActive.size()
			<< endl;
	cout << "size of nodeLocX:" << nodes.getInfoVecs().nodeLocX.size() << endl;
	cout << "size of nodeLocY:" << nodes.getInfoVecs().nodeLocY.size() << endl;
	cout << "size of nodeLocZ:" << nodes.getInfoVecs().nodeLocZ.size() << endl;
	cout << "size of nodeVelX:" << nodes.getInfoVecs().nodeVelX.size() << endl;
	cout << "size of nodeVelY:" << nodes.getInfoVecs().nodeVelY.size() << endl;
	cout << "size of nodeVelZ:" << nodes.getInfoVecs().nodeVelZ.size() << endl;
	cout << "size of CellType:" << nodes.getInfoVecs().nodeCellType.size()
			<< endl;
	cout << "size of nodeCellRank:" << nodes.getInfoVecs().nodeCellRank.size()
			<< endl;

	cout << "start position of Profile is "
			<< nodes.getAllocPara().startPosProfile << endl;
	cout << "start position of ECM is " << nodes.getAllocPara().startPosECM
			<< endl;
	cout << "start position of Cells is " << nodes.getAllocPara().startPosCells
			<< endl;

	cout << "max node of one cell is " << nodes.getAllocPara().maxNodeOfOneCell
			<< endl;
	cout << "max number of cells is " << nodes.getAllocPara().maxCellCount
			<< endl;
	cout << "max total cell node count is "
			<< nodes.getAllocPara().maxTotalCellNodeCount << endl;
	cout << "current active cell count is "
			<< nodes.getAllocPara().currentActiveCellCount << endl;

	cout << "max node of one ECM is " << nodes.getAllocPara().maxNodePerECM
			<< endl;
	cout << "max number of ECm is " << nodes.getAllocPara().maxECMCount << endl;
	cout << "max total ECM node count is "
			<< nodes.getAllocPara().maxTotalECMNodeCount << endl;
	cout << "current active ECM count is "
			<< nodes.getAllocPara().currentActiveECM << endl;

	cout << "max profile node count is "
			<< nodes.getAllocPara().maxProfileNodeCount << endl;
	cout << "current active profile node count is "
			<< nodes.getAllocPara().currentActiveProfileNodeCount << endl;
//int jj;
//cin >> jj;
}


void SimulationDomainGPU::outputLabelMatrix(std::string resultNameBase,
		int rank, PixelizePara& pixelPara) {
	std::stringstream ss;
	ss << std::setw(5) << std::setfill('0') << rank;
	std::string resultNameRank = ss.str();
	std::string matrixFileName = resultNameBase + resultNameRank + ".dat";
	vector<vector<int> > matrix = nodes.obtainLabelMatrix(pixelPara);
	printMatrixToFile(matrix, matrixFileName);
}
