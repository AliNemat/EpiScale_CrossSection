#include "hip/hip_runtime.h"
/**
 * @file SimulationDomainGPU.cu
 * @brief this file contains domain level logic.
 * @author Wenzhao Sun wsun2@nd.edu
 * @bug no know bugs
 */

#include "SimulationDomainGPU.h"

using namespace std;

/**
 * Constructor.
 * reads values from config file.
 */
SimulationDomainGPU::SimulationDomainGPU() {
	readAllParameters();
	if (memPara.simuType == Beak) {
		initializeGrowthMap();
	}
}

void SimulationDomainGPU::initializeNodes(CartPara &cartPara,
		std::vector<SceNodeType>& cellTypes,
		std::vector<uint>& numOfInitActiveNodesOfCells,
		std::vector<CVector>& initBdryNodeVec,
		std::vector<CVector>& initProfileNodeVec,
		std::vector<CVector>& initCartNodeVec,
		std::vector<CVector>& initECMNodeVec,
		std::vector<CVector>& initFNMNodeVec,
		std::vector<CVector>& initMXNodeVec) {

	/*
	 * number of boundary nodes is always fixed.
	 */
	uint bdryNodeCount = initBdryNodeVec.size();
	/*
	 * total potential number of profile nodes could be more than initial number provided,
	 * because the total length of profile might increase.
	 * FinalToInitProfileNodeCountRatio is defined in Config file.
	 */
	uint maxProfileNodeCount = initProfileNodeVec.size()
			* memPara.FinalToInitProfileNodeCountRatio;

	/**
	 * different from profile nodes, the cartilage nodes should already
	 * have buffer before executing this method.
	 */
	uint maxCartNodeCount = initCartNodeVec.size();

	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(bdryNodeCount, maxProfileNodeCount, maxCartNodeCount,
			memPara.maxECMInDomain, memPara.maxNodePerECM,
			memPara.maxCellInDomain, memPara.maxNodePerCell, memPara.isStab);

	//cartilage.distributeIsActive();
	//cartilage.initializeNodes(initCartNodeVec);

	/*
	 * first step: error checking.
	 * we need to first check if inputs are valid
	 */
	// get max node per cell. should be defined previously.
	uint maxNodePerCell = nodes.getAllocPara().maxNodeOfOneCell;
	// get max node per ECM. Should be defined previously.
	uint maxNodePerECM = nodes.getAllocPara().maxNodePerECM;
	// check if we successfully loaded maxNodePerCell
	assert(maxNodePerCell != 0);

	// obtain sizes of the input arrays
	//uint bdryNodeCount = initBdryNodeVec.size();
	uint ProfileNodeCount = initProfileNodeVec.size();
	uint CartNodeCount = initCartNodeVec.size();
	uint ECMNodeCount = initECMNodeVec.size();
	uint FNMNodeCount = initFNMNodeVec.size();
	uint MXNodeCount = initMXNodeVec.size();

	// array size of cell type array
	uint cellTypeSize = cellTypes.size();
	// array size of initial active node count of cells array.
	uint initNodeCountSize = numOfInitActiveNodesOfCells.size();
	// two sizes must match.
	assert(cellTypeSize == initNodeCountSize);

	// size of inputs must be divided exactly by max node per cell.
	// uint bdryRemainder = bdryNodeCountX % maxNodePerCell;
	uint ecmRemainder = 0;
	uint ecmQuotient = 0;
	if (memPara.simuType == Beak) {
		ecmQuotient = ECMNodeCount / maxNodePerECM;
		ecmRemainder = ECMNodeCount % maxNodePerECM;
	}
	uint fnmRemainder = FNMNodeCount % maxNodePerCell;
	uint mxRemainder = MXNodeCount % maxNodePerCell;

	// uint bdryQuotient = bdryNodeCountX / maxNodePerCell;
	uint fnmQuotient = FNMNodeCount / maxNodePerCell;
	uint mxQuotient = MXNodeCount / maxNodePerCell;

	// remainder must be zero.
	if ((fnmRemainder != 0) || (mxRemainder != 0) || (ecmRemainder != 0)) {
		throw SceException("Initialization vector size incorrect!",
				InputInitException);
	}
	// size of cellType array and sum of all cell types must match.
	assert(fnmQuotient + mxQuotient == cellTypeSize);

	for (uint i = 0; i < cellTypeSize; i++) {
		if (i < fnmQuotient) {
			assert(cellTypes[i] == FNM);
		} else {
			assert(cellTypes[i] == MX);
		}
	}
	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */
	NodeAllocPara para = nodes.getAllocPara();
	para.currentActiveCellCount = fnmQuotient + mxQuotient;
	para.currentActiveECM = ecmQuotient;
	para.currentActiveProfileNodeCount = ProfileNodeCount;
	nodes.setAllocPara(para);

	assert(nodes.getAllocPara().startPosProfile == bdryNodeCount);

	nodes.initValues(initBdryNodeVec, initProfileNodeVec, initCartNodeVec,
			initECMNodeVec, initFNMNodeVec, initMXNodeVec);

	/**
	 * setting the cartilage related parameters in the simulation domain.
	 */
	if (memPara.simuType == Beak && !memPara.isStab) {
		cartilage.setCartPara(cartPara);
		cartilage.initializeMem(&nodes);
	}

	cells = SceCells(&nodes, numOfInitActiveNodesOfCells, cellTypes);
}

void SimulationDomainGPU::initializeNodes_M(std::vector<SceNodeType>& nodeTypes,
		std::vector<uint>& numOfInitActiveEpiNodeCounts,
		std::vector<uint>& numOfInitActiveInternalNodeCounts,
		std::vector<CVector>& initNodesVec) {
	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(0, 0, 0, 0, 0, memPara.maxCellInDomain,
			memPara.maxAllNodePerCell, memPara.isStab);

	// array size of cell type array
	uint nodeTypeSize = nodeTypes.size();
	// array size of initial active node count of cells array.
	uint initEpiNodeCountSize = numOfInitActiveEpiNodeCounts.size();
	uint initInternalNodeCountSize = numOfInitActiveInternalNodeCounts.size();
	// two sizes must match.
	assert(initEpiNodeCountSize == initInternalNodeCountSize);
	assert(initNodesVec.size() == nodeTypes.size());

	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */
	NodeAllocPara_M para = nodes.getAllocParaM();
	para.currentActiveCellCount = initNodesVec.size() / para.maxAllNodePerCell;
	nodes.setAllocParaM(para);

	std::vector<CVector> dummyEmptyPos;
	nodes.initValues_M(dummyEmptyPos, initNodesVec, nodeTypes);

	cells = SceCells(&nodes, numOfInitActiveEpiNodeCounts,
			numOfInitActiveInternalNodeCounts);
}

void SimulationDomainGPU::initialize_v2(SimulationInitData_V2& initData) {
	std::cout << "begin initialization process" << std::endl;
	memPara.isStab = initData.isStab;
	initializeNodes(initData.cartPara, initData.cellTypes,
			initData.numOfInitActiveNodesOfCells, initData.initBdryNodeVec,
			initData.initProfileNodeVec, initData.initCartNodeVec,
			initData.initECMNodeVec, initData.initFNMNodeVec,
			initData.initMXNodeVec);
	std::cout << "finished init simulation domain nodes" << std::endl;
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
	std::cout << "finished init nodes dimension" << std::endl;
	// The domain task is not stabilization unless specified in the next steps.
	stabPara.isProcessStab = false;
}

void SimulationDomainGPU::initialize_v2_M(SimulationInitData_V2_M& initData) {
	std::cout << "begin initialization process" << std::endl;
	memPara.isStab = initData.isStab;
	CartPara dummyCart;
	initializeNodes_M(initData.nodeTypes, initData.InitActiveEpiNodePerCellArr,
			initData.InitActiveInternalNodePerCellArr, initData.initNodeVec);
	std::cout << "finished init simulation domain nodes" << std::endl;
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
	std::cout << "finished init nodes dimension" << std::endl;
	// The domain task is not stabilization unless specified in the next steps.
	stabPara.isProcessStab = false;
}

/**
 * Highest level logic of domain.
 *
 */
void SimulationDomainGPU::runAllLogic(double dt) {
	if (memPara.simuType == Beak && !stabPara.isProcessStab) {
		nodes.processCartGrowthDir(cartilage.getCartPara().growthDir);
	}

	if (memPara.simuType == Beak) {
		nodes.calculateAndApplySceForces();
	} else if (memPara.simuType == Disc) {
		nodes.sceForcesDisc();
	}

	// This function only calculates velocity.

	// Only beak simulation need to take care of cartilage.
	if (memPara.simuType == Beak && !stabPara.isProcessStab) {
		// cartilage logics must come before cell logics, because node velocities will be modified
		// in cell logic and consequently we won't be able to compute cartilage data.
		// also responsible for handling interaction between epithelium layer and carilage.
		cartilage.runAllLogics(dt);
	}

	// This function applies velocity so nodes actually move inside this function.
	if (memPara.simuType == Beak) {
		cells.runAllCellLevelLogicsBeak(dt, growthMap, growthMap2);
	} else if (memPara.simuType == Disc) {
		cells.runAllCellLevelLogicsDisc(dt);
	}

	if (memPara.simuType == SingleCellTest) {
		nodes.sceForcesDisc();
		cells.runStretchTest(dt);
	}
}

void SimulationDomainGPU::readMemPara() {
	int simuTypeConfigValue =
			globalConfigVars.getConfigValue("SimulationType").toInt();

	memPara.simuType = parseTypeFromConfig(simuTypeConfigValue);

	memPara.maxCellInDomain =
			globalConfigVars.getConfigValue("MaxCellInDomain").toInt();
	memPara.maxNodePerCell =
			globalConfigVars.getConfigValue("MaxNodePerCell").toInt();
	if (memPara.simuType == Beak) {
		memPara.maxECMInDomain = globalConfigVars.getConfigValue(
				"MaxECMInDomain").toInt();
		memPara.maxNodePerECM =
				globalConfigVars.getConfigValue("MaxNodePerECM").toInt();
		memPara.FinalToInitProfileNodeCountRatio =
				globalConfigVars.getConfigValue(
						"FinalToInitProfileNodeCountRatio").toDouble();
		//memPara.FinalToInitCartNodeCountRatio = globalConfigVars.getConfigValue(
		//		"FinalToInitCartNodeCountRatio").toDouble();
	} else {
		memPara.maxECMInDomain = 0;
		memPara.maxNodePerECM = 0;
		memPara.FinalToInitProfileNodeCountRatio = 0;
	}

	if (memPara.simuType == Disc_M) {
		memPara.maxEpiNodePerCell = globalConfigVars.getConfigValue(
				"MaxEpiNodePerCell").toInt();
		memPara.maxInternalNodePerCell = globalConfigVars.getConfigValue(
				"MaxInternalNodePerCell").toInt();
		memPara.maxAllNodePerCell = memPara.maxEpiNodePerCell
				+ memPara.maxInternalNodePerCell;
	}
}

void SimulationDomainGPU::readDomainPara() {
	domainPara.minX = globalConfigVars.getConfigValue("DOMAIN_XMIN").toDouble();
	domainPara.maxX = globalConfigVars.getConfigValue("DOMAIN_XMAX").toDouble();
	domainPara.minY = globalConfigVars.getConfigValue("DOMAIN_YMIN").toDouble();
	domainPara.maxY = globalConfigVars.getConfigValue("DOMAIN_YMAX").toDouble();
	domainPara.minZ = globalConfigVars.getConfigValue("DOMAIN_ZMIN").toDouble();
	domainPara.maxZ = globalConfigVars.getConfigValue("DOMAIN_ZMAX").toDouble();
	domainPara.gridSpacing = nodes.getMaxEffectiveRange();
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInYDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
}

void SimulationDomainGPU::readChemPara() {
	chemPara.growthGridXDim =
			globalConfigVars.getConfigValue("GrowthGridXDim").toInt();
	chemPara.growthGridYDim =
			globalConfigVars.getConfigValue("GrowthGridYDim").toInt();
	chemPara.growthGridSpacing = globalConfigVars.getConfigValue(
			"GrowthGridSpacing").toDouble();
	chemPara.growthGridLowerLeftPtX = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtX").toDouble();
	chemPara.growthGridLowerLeftPtY = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtY").toDouble();

	chemPara.growthMorCenterXCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoord").toDouble();
	chemPara.growthMorCenterYCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoord").toDouble();
	chemPara.growthMorHighConcen = globalConfigVars.getConfigValue(
			"GrowthMorHighConcen").toDouble();
	chemPara.growthMorLowConcen = globalConfigVars.getConfigValue(
			"GrowthMorLowConcen").toDouble();
	chemPara.growthMorDiffSlope = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlope").toDouble();

	chemPara.growthMorCenterXCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoordMX").toDouble();
	chemPara.growthMorCenterYCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoordMX").toDouble();
	chemPara.growthMorHighConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorHighConcenMX").toDouble();
	chemPara.growthMorLowConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorLowConcenMX").toDouble();
	chemPara.growthMorDiffSlopeMX = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlopeMX").toDouble();
}

void SimulationDomainGPU::readAllParameters() {
	readMemPara();
	readDomainPara();
	if (memPara.simuType == Beak) {
		readChemPara();
	}
}

void SimulationDomainGPU::initializeGrowthMap() {
	growthMap = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoord,
			chemPara.growthMorCenterYCoord, chemPara.growthMorHighConcen,
			chemPara.growthMorLowConcen, chemPara.growthMorDiffSlope);

	//cout << "after created growthMap1" << endl;
	growthMap2 = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap2.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoordMX,
			chemPara.growthMorCenterYCoordMX, chemPara.growthMorHighConcenMX,
			chemPara.growthMorLowConcenMX, chemPara.growthMorDiffSlopeMX);
	//cout << "after created growthMap2" << endl;
}

std::vector<CVector> SimulationDomainGPU::stablizeCellCenters(
		SimulationInitData_V2 &initData) {

	std::vector<CVector> result;

	stabPara.outputFrameCount = globalConfigVars.getConfigValue(
			"StabFrameCount").toInt();
	stabPara.totalIterCount = globalConfigVars.getConfigValue(
			"StabTotalIterCount").toInt();
	stabPara.bdrySpacingRatio = globalConfigVars.getConfigValue(
			"StabBdrySpacingRatio").toDouble();
	stabPara.dt = globalConfigVars.getConfigValue("StabDt").toDouble();
	stabPara.outputAniName =
			globalConfigVars.getConfigValue("StabAniName").toString();

	initialize_v2(initData);
	stabPara.isProcessStab = true;
	int aniAuxPara;
	if (stabPara.outputFrameCount == 0) {
		aniAuxPara = INT_MAX;
	} else {
		aniAuxPara = (double) (stabPara.totalIterCount)
				/ stabPara.outputFrameCount;
	}

	AnimationCriteria aniCri;
	aniCri.defaultEffectiveDistance = globalConfigVars.getConfigValue(
			"IntraLinkDisplayRange").toDouble();
	int configAniType =
			globalConfigVars.getConfigValue("AnimationType").toInt();
	aniCri.animationType = CellType;

	uint index = 0;
	for (int i = 0; i < stabPara.totalIterCount; i++) {
		//std::cout << "in stablizing, before run all logics" << std::endl;
		if (i % aniAuxPara == 0) {
			outputVtkFilesWithCri(stabPara.outputAniName, index, aniCri);
			index++;
		}
		runAllLogic(stabPara.dt);
	}

	result = cells.getAllCellCenters();

	cout << "finished stablizeCellCenters" << endl;
	cout.flush();
	return result;
}

void SimulationDomainGPU::outputVtkFilesWithCri(std::string scriptNameBase,
		int rank, AnimationCriteria aniCri) {
	nodes.prepareSceForceComputation();
	VtkAnimationData aniData = nodes.obtainAnimationData(aniCri);
	aniData.outputVtkAni(scriptNameBase, rank);
}

void SimulationDomainGPU::printDomainInformation() {
	cout << "Begin output information about nodes:" << endl;
	cout << "size of isActive:" << nodes.getInfoVecs().nodeIsActive.size()
			<< endl;
	cout << "size of nodeLocX:" << nodes.getInfoVecs().nodeLocX.size() << endl;
	cout << "size of nodeLocY:" << nodes.getInfoVecs().nodeLocY.size() << endl;
	cout << "size of nodeLocZ:" << nodes.getInfoVecs().nodeLocZ.size() << endl;
	cout << "size of nodeVelX:" << nodes.getInfoVecs().nodeVelX.size() << endl;
	cout << "size of nodeVelY:" << nodes.getInfoVecs().nodeVelY.size() << endl;
	cout << "size of nodeVelZ:" << nodes.getInfoVecs().nodeVelZ.size() << endl;
	cout << "size of CellType:" << nodes.getInfoVecs().nodeCellType.size()
			<< endl;
	cout << "size of nodeCellRank:" << nodes.getInfoVecs().nodeCellRank.size()
			<< endl;

	cout << "start position of Profile is "
			<< nodes.getAllocPara().startPosProfile << endl;
	cout << "start position of ECM is " << nodes.getAllocPara().startPosECM
			<< endl;
	cout << "start position of Cells is " << nodes.getAllocPara().startPosCells
			<< endl;

	cout << "max node of one cell is " << nodes.getAllocPara().maxNodeOfOneCell
			<< endl;
	cout << "max number of cells is " << nodes.getAllocPara().maxCellCount
			<< endl;
	cout << "max total cell node count is "
			<< nodes.getAllocPara().maxTotalCellNodeCount << endl;
	cout << "current active cell count is "
			<< nodes.getAllocPara().currentActiveCellCount << endl;

	cout << "max node of one ECM is " << nodes.getAllocPara().maxNodePerECM
			<< endl;
	cout << "max number of ECm is " << nodes.getAllocPara().maxECMCount << endl;
	cout << "max total ECM node count is "
			<< nodes.getAllocPara().maxTotalECMNodeCount << endl;
	cout << "current active ECM count is "
			<< nodes.getAllocPara().currentActiveECM << endl;

	cout << "max profile node count is "
			<< nodes.getAllocPara().maxProfileNodeCount << endl;
	cout << "current active profile node count is "
			<< nodes.getAllocPara().currentActiveProfileNodeCount << endl;
}

vector<vector<int> > SimulationDomainGPU::outputLabelMatrix(
		std::string resultNameBase, int rank, PixelizePara& pixelPara) {
	std::stringstream ss;
	ss << std::setw(5) << std::setfill('0') << rank;
	std::string resultNameRank = ss.str();
	std::string matrixFileName = resultNameBase + resultNameRank + ".dat";
	vector<vector<int> > matrix = nodes.obtainLabelMatrix(pixelPara);
	printMatrixToFile(matrix, matrixFileName);
	return matrix;
}

void SimulationDomainGPU::outputGrowthProgressAuxFile(int step) {
	static bool isFirstTime = true;
	std::string auxDataFileName = globalConfigVars.getConfigValue(
			"DataOutputFolder").toString()
			+ globalConfigVars.getConfigValue("GrowthAuxFileName").toString();
	if (isFirstTime) {
		std::remove(auxDataFileName.c_str());
		isFirstTime = false;
	}
	std::cout << "Updating growth progress file" << std::endl;
	ofstream ofs;
	ofs.open(auxDataFileName.c_str(), ios::app);
	ofs << step << " ";
	std::vector<double> growProVec = cells.getGrowthProgressVec();
	for (std::vector<double>::iterator it = growProVec.begin();
			it != growProVec.end(); ++it) {
		ofs << *it << " ";
	}
	ofs << std::endl;
	ofs.close();
}

void SimulationDomainGPU::analyzeLabelMatrix(vector<vector<int> > &labelMatrix,
		int step, std::string &imageFileNameBase, std::string &statFileName) {
	ResAnalysisHelper resHelper;

	std::stringstream ss;
	ss << std::setw(5) << std::setfill('0') << step;
	std::string imgNameRank = ss.str();
	std::string imgFileName = imageFileNameBase + imgNameRank + ".bmp";

	resHelper.outputImg_formatBMP(imgFileName, labelMatrix);
	std::vector<double> growthProVec = cells.getGrowthProgressVec();
	if (memPara.simuType == Disc) {
		resHelper.outputStat_PolygonCounting(statFileName, step, labelMatrix,
				growthProVec);
		outputGrowthProgressAuxFile(step);
	} else {
		resHelper.outputStat_PolygonCounting(statFileName, step, labelMatrix);
	}
}

void SimulationDomainGPU::performAblation(AblationEvent& ablEvent) {
	thrust::host_vector<double> xCoord = nodes.getInfoVecs().nodeLocX;
	thrust::host_vector<double> yCoord = nodes.getInfoVecs().nodeLocY;

	AblationEvent aa;

	for (uint i = 0; i < xCoord.size(); i++) {
		double xDiff = xCoord[i] - 25.3;
		double yDiff = yCoord[i] - 25.2;
		if (xDiff * xDiff + yDiff * yDiff < 0.04) {
			uint cellRank = i / 90;
			uint nodeRank = i % 90;
			std::cout << "cell : " << cellRank << ", node: " << nodeRank
					<< "pos: (" << xCoord[i] << "," << yCoord[i] << ")"
					<< std::endl;
			bool found = false;
			for (uint j = 0; j < aa.ablationCells.size(); j++) {
				if (aa.ablationCells[j].cellNum == cellRank) {
					found = true;
					aa.ablationCells[j].nodeNums.push_back(nodeRank);
				}
			}
			if (!found) {
				AblaInfo cellNew;
				cellNew.cellNum = cellRank;
				cellNew.nodeNums.push_back(nodeRank);
				aa.ablationCells.push_back(cellNew);
			}
		}
	}

	aa.printInfo();
	int jj;
	cin >> jj;

	cells.runAblationTest(aa);
}
