#include "hip/hip_runtime.h"
/**
 * @file SimulationDomainGPU.cu
 * @brief this file contains domain level logic.
 * @author Wenzhao Sun wsun2@nd.edu
 * @bug no know bugs
 */

#include "SimulationDomainGPU.h"

using namespace std;

/**
 * Constructor.
 * reads values from config file.
 */
SimulationDomainGPU::SimulationDomainGPU() {
	readAllParameters();
	initializeGrowthMap();
}

/**
 * Initialize five different cell nodes.
 * We have to initialize five types of cells:
 * first is Boundary (B), fixed nodes on the boundary;
 * second is Profile (P), Epithilum cells;
 * third is ECM (E), extra-cellular matrix;
 * fourth is FNM (F), front nasal mass;
 * fifth is MX (M) maxillary cells.
 *
 * like this:
 * B-B-B-B-B-B-B-B-B-P-P-P-P-E-E-E-E-F-F-F-F-F-F-F-F-M-M-M-M-M-M-M-M
 * B, P and E is fixed. F and M will grow.
 * Rules:
 * 1a, Number of boundary nodes is fixed.
 * 1b, Profile nodes may or may not increase. Still testing the model.
 *     however the space is always reserved for it, though some spaces may not be active.
 * 1c, Extra-cellular matrix will grow.
 *     however the space is always reserved for it, but some spaces are not active.
 * 1d, In F part, each input vector must be divided exactly by (max node per cell)
 * 1e, In M part, each input vector must be divided exactly by (max node per cell)
 * 2a, Sum of number of cells from init FNM and init MX input vectors must be size of cellTypes
 *     so that all cells will have its own type
 * 2b, Read number of node per cell, etc, from config file.
 * 3a, First part of this function is error checking.
 * 3b, Second part of this function is the actual initialization
 */
void SimulationDomainGPU::initialCellsOfFiveTypes(
		std::vector<SceNodeType> &cellTypes,
		std::vector<uint> &numOfInitActiveNodesOfCells,
		std::vector<double> &initBdryCellNodePosX,
		std::vector<double> &initBdryCellNodePosY,
		std::vector<double> &initProfileNodePosX,
		std::vector<double> &initProfileNodePosY,
		std::vector<double> &initECMNodePosX,
		std::vector<double> &initECMNodePosY,
		std::vector<double> &initFNMCellNodePosX,
		std::vector<double> &initFNMCellNodePosY,
		std::vector<double> &initMXCellNodePosX,
		std::vector<double> &initMXCellNodePosY) {

	/*
	 * zero step: redefine nodes.
	 */
	/*
	 * number of boundary nodes is always fixed.
	 */
	uint bdryNodeCount = initBdryCellNodePosX.size();
	// total potential number of profile nodes could be more than initial number provided,
	// because the total length of profile might increase.
	// FinalToInitProfileNodeCountRatio is defined in Config file.
	uint maxProfileNodeCount = initProfileNodePosX.size()
			* memPara.FinalToInitProfileNodeCountRatio;

	/*
	 * Initialize SceNodes by constructor. first two parameters come from input parameters
	 * while the last four parameters come from Config file.
	 */
	nodes = SceNodes(bdryNodeCount, maxProfileNodeCount, memPara.maxECMInDomain,
			memPara.maxNodePerECM, memPara.maxCellInDomain,
			memPara.maxNodePerCell);
	/*
	 * Initialize SceCells_M ( M means modified) by nodes information.
	 */
	cells_m = SceCells_M(&nodes);

	/*
	 * first step: error checking.
	 * we need to first check if inputs are valid
	 */
	cout << "begin init cells of five types" << endl;
	// get max node per cell. should be defined previously.
	uint maxNodePerCell = nodes.maxNodeOfOneCell;
	// get max node per ECM. Should be defined previously.
	uint maxNodePerECM = nodes.getMaxNodePerEcm();
	// check if we successfully loaded maxNodePerCell
	assert(maxNodePerCell != 0);

	// obtain sizes of the input arrays
	uint bdryNodeCountX = initBdryCellNodePosX.size();
	uint bdryNodeCountY = initBdryCellNodePosY.size();
	uint ProfileNodeCountX = initProfileNodePosX.size();
	uint ProfileNodeCountY = initProfileNodePosY.size();
	uint ECMNodeCountX = initECMNodePosX.size();
	uint ECMNodeCountY = initECMNodePosY.size();
	uint FNMNodeCountX = initFNMCellNodePosX.size();
	uint FNMNodeCountY = initFNMCellNodePosY.size();
	uint MXNodeCountX = initMXCellNodePosX.size();
	uint MXNodeCountY = initMXCellNodePosY.size();

	cout << "size of all node vectors: Boundary = " << bdryNodeCountX
			<< ", init profile node count =" << "," << ProfileNodeCountX
			<< "max profile node count = " << maxProfileNodeCount
			<< ", initial ECM node count = " << ECMNodeCountX
			<< ", init FNM node count = " << FNMNodeCountX
			<< ", init MX node count = " << MXNodeCountX << endl;

	// array size of cell type array
	uint cellTypeSize = cellTypes.size();
	// array size of initial active node count of cells array.
	uint initNodeCountSize = numOfInitActiveNodesOfCells.size();
	// two sizes must match.
	assert(cellTypeSize == initNodeCountSize);
	// size of X and Y must match.
	assert(bdryNodeCountX == bdryNodeCountY);
	assert(ECMNodeCountX == ECMNodeCountY);
	assert(ProfileNodeCountX == ProfileNodeCountY);
	assert(FNMNodeCountX == FNMNodeCountY);
	assert(MXNodeCountX == MXNodeCountY);

	cout << "passed init checks" << endl;

	// size of inputs must be divided exactly by max node per cell.
	// uint bdryRemainder = bdryNodeCountX % maxNodePerCell;
	uint ecmRemainder = ECMNodeCountX % maxNodePerECM;
	uint fnmRemainder = FNMNodeCountX % maxNodePerCell;
	uint mxRemainder = MXNodeCountX % maxNodePerCell;

	// uint bdryQuotient = bdryNodeCountX / maxNodePerCell;
	uint ecmQuotient = ECMNodeCountX / maxNodePerECM;
	uint fnmQuotient = FNMNodeCountX / maxNodePerCell;
	uint mxQuotient = MXNodeCountX / maxNodePerCell;

	// for now we try to make boundary cells one complete part so ....
	//uint bdryRemainder = 0;
	uint bdryQuotient = 1;

	// for now we try to make profile nodes one complete part soremdiner = 0 and quotient = 1
	//uint profileRemainder = 0;
	uint profileQuotient = 1;

	// remainder must be zero.
	assert((fnmRemainder == 0) && (mxRemainder == 0) && (ecmRemainder == 0));
	// size of cellType array and sum of all cell types must match.
	assert(fnmQuotient + mxQuotient == cellTypeSize);

	cerr << "passed size assertion" << endl;

	// make sure the cell types follow format requirement.
	// must follow sequence : B - P - E - F - M
	int counter = 0;
	//SceNodeType cellTypesForEachLevel[5] = { Boundary, Profile, ECM, FNM, MX };
	int bounds[5];
	bounds[0] = bdryQuotient;
	bounds[1] = bounds[0] + profileQuotient;
	bounds[2] = bounds[1] + ecmQuotient;
	bounds[3] = bounds[2] + fnmQuotient;
	bounds[4] = bounds[3] + mxQuotient;
	int level = 0;
	while (counter < cellTypeSize) {
		// if count is already beyond the bound, we need to increase the current level.
		if (counter == bounds[level]) {
			level++;
		}
		// make sure that the input cell types array fits the calculated result.
		// depreciated -- requirement changed.
		// assert(cellTypes[counter] == cellTypesForEachLevel[level]);
		counter++;
	}
	cerr << "before set parameters" << endl;
	/*
	 * second part: actual initialization
	 * copy data from main system memory to GPU memory
	 */

	nodes.setCurrentActiveCellCount(fnmQuotient + mxQuotient);
	cells_m.currentActiveCellCount = fnmQuotient + mxQuotient;
	nodes.setCurrentActiveEcm(ecmQuotient);
	cells_m.currentActiveECMCount = ecmQuotient;

	nodes.currentActiveProfileNodeCount = ProfileNodeCountX;

	assert(nodes.startPosProfile == bdryNodeCountX);

	uint totalSize = nodes.nodeLocX.size();

	// set cell types
	thrust::device_vector<SceNodeType> cellTypesToPass = cellTypes;

	// copy initial active node count info to GPU
	thrust::copy(numOfInitActiveNodesOfCells.begin(),
			numOfInitActiveNodesOfCells.end(),
			cells_m.activeNodeCountOfThisCell.begin());

	nodes.initValues(initBdryCellNodePosX, initBdryCellNodePosY,
			initProfileNodePosX, initProfileNodePosY, initECMNodePosX,
			initECMNodePosY, initFNMCellNodePosX, initFNMCellNodePosY,
			initMXCellNodePosX, initMXCellNodePosY);

	// set cell types
	cells_m.setCellTypes(cellTypesToPass);
	cells_m.distributeIsActiveInfo();
}

void SimulationDomainGPU::initialize_V2(SimulationInitData &initData) {
	initialCellsOfFiveTypes(initData.cellTypes,
			initData.numOfInitActiveNodesOfCells, initData.initBdryCellNodePosX,
			initData.initBdryCellNodePosY, initData.initProfileNodePosX,
			initData.initProfileNodePosY, initData.initECMNodePosX,
			initData.initECMNodePosY, initData.initFNMCellNodePosX,
			initData.initFNMCellNodePosY, initData.initMXCellNodePosX,
			initData.initMXCellNodePosY);
	//cout << "finished init cells of five types" << endl;
	nodes.initDimension(domainPara.minX, domainPara.maxX, domainPara.minY,
			domainPara.maxY, domainPara.gridSpacing);
}

/**
 * Highest level logic of domain.
 *
 */
void SimulationDomainGPU::runAllLogic(double dt) {
	nodes.calculateAndApplySceForces();
	cells_m.runAllCellLevelLogics(dt, growthMap, growthMap2);
}

void SimulationDomainGPU::readMemPara() {
	memPara.maxCellInDomain =
			globalConfigVars.getConfigValue("MaxCellInDomain").toInt();
	memPara.maxNodePerCell =
			globalConfigVars.getConfigValue("MaxNodePerCell").toDouble();
	memPara.maxECMInDomain =
			globalConfigVars.getConfigValue("MaxECMInDomain").toDouble();
	memPara.maxNodePerECM =
			globalConfigVars.getConfigValue("MaxNodePerECM").toDouble();
	memPara.FinalToInitProfileNodeCountRatio = globalConfigVars.getConfigValue(
			"FinalToInitProfileNodeCountRatio").toDouble();
}

void SimulationDomainGPU::readDomainPara() {
	domainPara.minX = globalConfigVars.getConfigValue("DOMAIN_XMIN").toDouble();
	domainPara.maxX = globalConfigVars.getConfigValue("DOMAIN_XMAX").toDouble();
	domainPara.minY = globalConfigVars.getConfigValue("DOMAIN_YMIN").toDouble();
	domainPara.maxY = globalConfigVars.getConfigValue("DOMAIN_YMAX").toDouble();
	domainPara.minZ = globalConfigVars.getConfigValue("DOMAIN_ZMIN").toDouble();
	domainPara.maxZ = globalConfigVars.getConfigValue("DOMAIN_ZMAX").toDouble();
	domainPara.gridSpacing = globalConfigVars.getConfigValue(
			"Cell_Center_Interval").toDouble();
	domainPara.numOfBucketsInXDim = (domainPara.maxX - domainPara.minX)
			/ domainPara.gridSpacing + 1;
	domainPara.numOfBucketsInXDim = (domainPara.maxY - domainPara.minY)
			/ domainPara.gridSpacing + 1;
}

void SimulationDomainGPU::readChemPara() {
	chemPara.growthGridXDim =
			globalConfigVars.getConfigValue("GrowthGridXDim").toInt();
	chemPara.growthGridYDim =
			globalConfigVars.getConfigValue("GrowthGridYDim").toInt();
	chemPara.growthGridSpacing = globalConfigVars.getConfigValue(
			"GrowthGridSpacing").toDouble();
	chemPara.growthGridLowerLeftPtX = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtX").toDouble();
	chemPara.growthGridLowerLeftPtY = globalConfigVars.getConfigValue(
			"GrowthGridLowerLeftPtY").toDouble();

	chemPara.growthMorCenterXCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoord").toDouble();
	chemPara.growthMorCenterYCoord = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoord").toDouble();
	chemPara.growthMorHighConcen = globalConfigVars.getConfigValue(
			"GrowthMorHighConcen").toDouble();
	chemPara.growthMorLowConcen = globalConfigVars.getConfigValue(
			"GrowthMorLowConcen").toDouble();
	chemPara.growthMorDiffSlope = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlope").toDouble();

	chemPara.growthMorCenterXCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterXCoordMX").toDouble();
	chemPara.growthMorCenterYCoordMX = globalConfigVars.getConfigValue(
			"GrowthMorCenterYCoordMX").toDouble();
	chemPara.growthMorHighConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorHighConcenMX").toDouble();
	chemPara.growthMorLowConcenMX = globalConfigVars.getConfigValue(
			"GrowthMorLowConcenMX").toDouble();
	chemPara.growthMorDiffSlopeMX = globalConfigVars.getConfigValue(
			"GrowthMorDiffSlopeMX").toDouble();
}

void SimulationDomainGPU::readAllParameters() {
	readMemPara();
	readDomainPara();
	readChemPara();
}

void SimulationDomainGPU::initializeGrowthMap() {
	growthMap = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoord,
			chemPara.growthMorCenterYCoord, chemPara.growthMorHighConcen,
			chemPara.growthMorLowConcen, chemPara.growthMorDiffSlope);

	//cout << "after created growthMap1" << endl;
	growthMap2 = GrowthDistriMap(chemPara.growthGridXDim,
			chemPara.growthGridYDim, chemPara.growthGridSpacing);
	growthMap2.initialize(chemPara.growthGridLowerLeftPtX,
			chemPara.growthGridLowerLeftPtY, chemPara.growthMorCenterXCoordMX,
			chemPara.growthMorCenterYCoordMX, chemPara.growthMorHighConcenMX,
			chemPara.growthMorLowConcenMX, chemPara.growthMorDiffSlopeMX);
	//cout << "after created growthMap2" << endl;
}

void SimulationDomainGPU::outputVtkFilesWithColor_v3(std::string scriptNameBase,
		int rank, AnimationCriteria aniCri) {
	nodes.prepareSceForceComputation();
	VtkAnimationData aniData = nodes.obtainAnimationData(aniCri);
	aniData.outputVtkAni(scriptNameBase, rank);
}

void SimulationDomainGPU::checkIfAllDataFieldsValid() {
	cout << "Begin output information about nodes:" << endl;
	cout << "size of isActive:" << nodes.nodeIsActive.size() << endl;
	cout << "size of nodeLocX:" << nodes.nodeLocX.size() << endl;
	cout << "size of nodeLocY:" << nodes.nodeLocY.size() << endl;
	cout << "size of nodeLocZ:" << nodes.nodeLocZ.size() << endl;
	cout << "size of nodeVelX:" << nodes.nodeVelX.size() << endl;
	cout << "size of nodeVelY:" << nodes.nodeVelY.size() << endl;
	cout << "size of nodeVelZ:" << nodes.nodeVelZ.size() << endl;
	cout << "size of CellType:" << nodes.nodeCellType.size() << endl;
	cout << "size of nodeCellRank:" << nodes.nodeCellRank.size() << endl;

	cout << "start position of Profile is " << nodes.startPosProfile << endl;
	cout << "start position of ECM is " << nodes.startPosECM << endl;
	cout << "start position of Cells is " << nodes.startPosCells << endl;

	cout << "max node of one cell is " << nodes.maxNodeOfOneCell << endl;
	cout << "max number of cells is " << nodes.maxCellCount << endl;
	cout << "max total cell node count is " << nodes.maxTotalCellNodeCount
			<< endl;
	cout << "current active cell count is " << nodes.currentActiveCellCount
			<< endl;

	cout << "max node of one ECM is " << nodes.maxNodePerECM << endl;
	cout << "max number of ECm is " << nodes.maxECMCount << endl;
	cout << "max total ECM node count is " << nodes.maxTotalECMNodeCount
			<< endl;
	cout << "current active ECM count is " << nodes.currentActiveECM << endl;

	cout << "max profile node count is " << nodes.maxProfileNodeCount << endl;
	cout << "current active profile node count is "
			<< nodes.currentActiveProfileNodeCount << endl;
	//int jj;
	//cin >> jj;
}

